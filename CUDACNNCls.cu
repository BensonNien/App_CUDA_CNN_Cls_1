#include "hip/hip_runtime.h"
/******************************************************************************
Date:  2022/09
Author: CHU-MIN, NIEN
Description: CUDA ver.
******************************************************************************/
#include <algorithm>

#include "CUDACNNCls.cuh"
#include "CUDACNNLayer.cuh"

// CUDA_Algo_Lib::CUDACNN

#define DERIV_ACTIVE_RELU(S) 1 // derivative of the relu as a function of the relu's output
namespace CUDA_Algo_Lib
{
	size_t g_idx_epoch = 0;//index of epoch
	size_t g_idx_itor = 0;//index of iterator
	size_t g_idx_iter_init_bias = 0;//index of iterator for initialize bias
	size_t g_idx_iteration_num = 0;//index of iteration
	size_t g_iteration_num = 0;//number of g_iteration_num
}

void CUDA_Algo_Lib::CUDACNN::Train(CUDA_Algo_Lib::DatasetLoadingParamPKG& r_dataset_param)
{
	std::cout << "Start train" << std::endl;

	CUDA_Algo_Lib::g_iteration_num = r_dataset_param.total_num_images_ / batch_size_;
	if ((r_dataset_param.total_num_images_ % batch_size_) != 0)
	{
		std::cout << "Please reset CUDA_Algo_Lib::CUDACNN::batch_size_!" << std::endl;
	}

	float* p_train_batch_data = nullptr;
	float* p_train_batch_label = nullptr;
	std::vector<float> vec_train_batch_data;
	std::vector<float> vec_train_batch_label;
	vec_train_batch_data.reserve(batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_);
	vec_train_batch_data.resize(batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_);
	vec_train_batch_label.reserve(batch_size_ * r_dataset_param.num_output_cls_);
	vec_train_batch_label.resize(batch_size_ * r_dataset_param.num_output_cls_);

	for (CUDA_Algo_Lib::g_idx_iteration_num = 0; CUDA_Algo_Lib::g_idx_iteration_num < CUDA_Algo_Lib::g_iteration_num; CUDA_Algo_Lib::g_idx_iteration_num++)
	{
		std::cout << "NO.of iteration(training): " << CUDA_Algo_Lib::g_idx_iteration_num << std::endl;
		size_t idx_loaded_dataset_batch = CUDA_Algo_Lib::g_idx_iteration_num % (r_dataset_param.total_num_images_ / batch_size_);
		for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
		{
			std::cout << "NO.of batch(training): " << idx_batch << std::endl;

			std::vector<float>::iterator shift_begin_iter_loaded_dataset_batch_data;
			std::vector<float>::iterator shift_begin_iter_loaded_dataset_batch_label;
			shift_begin_iter_loaded_dataset_batch_data = r_dataset_param.vec_images_.begin() + (idx_loaded_dataset_batch * batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_);
			shift_begin_iter_loaded_dataset_batch_label = r_dataset_param.vec_labels_.begin() + (idx_loaded_dataset_batch * batch_size_ * r_dataset_param.num_output_cls_);
			vec_train_batch_data.assign(shift_begin_iter_loaded_dataset_batch_data, (shift_begin_iter_loaded_dataset_batch_data + (batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_)));
			vec_train_batch_label.assign(shift_begin_iter_loaded_dataset_batch_label, (shift_begin_iter_loaded_dataset_batch_label + (batch_size_ * r_dataset_param.num_output_cls_)));

		}


		Forward(vec_train_batch_data.data());
		BackPropagation(vec_train_batch_data.data(), vec_train_batch_label.data());
		UpdateParas();


	}
	std::cout << "Finish train" << std::endl;

}

void CUDA_Algo_Lib::CUDACNN::Setup(size_t batch_size)
{
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.begin();

	(*iter).InitOutputMaps(batch_size);
	iter++;
	for (iter; iter < vec_layers_.end(); iter++)
	{
		CUDA_Algo_Lib::g_idx_iter_init_bias = CUDA_Algo_Lib::g_idx_iter_init_bias + 1;

		size_t frontMapNum = (*(iter - 1)).GetOutMapNum();

		switch ((*iter).GetType())
		{
		case 'I':
			break;
		case 'C':
			// set map RectSize
			(*iter).SetMapSize((*(iter - 1)).GetMapSize().Substract((*iter).GetKernelSize(), 1));
			// initial convolution kernel_, quantities: frontMapNum*outMapNum_
			(*iter).InitKernel(frontMapNum);
			(*iter).InitLastStepDeltaKernel(frontMapNum);//for adding momentum
			//each map has one bias_, so frontMapNum is not necessary
			(*iter).InitBias(frontMapNum, CUDA_Algo_Lib::g_idx_iter_init_bias);
			(*iter).InitErros(batch_size);
			// each r_layer should initialize output map
			(*iter).InitOutputMaps(batch_size);
			break;
		case 'S':
			(*iter).SetOutMapNum((frontMapNum));
			(*iter).SetMapSize((*(iter - 1)).GetMapSize().Divide((*iter).GetScaleSize()));
			(*iter).InitErros(batch_size);
			(*iter).InitOutputMaps(batch_size);
			break;
		case 'H':
			(*iter).InitOutputKernel(frontMapNum, (*(iter - 1)).GetMapSize());
			(*iter).InitOutputLastStepDeltaKernel(frontMapNum, (*(iter - 1)).GetMapSize());//for adding momentum			
			(*iter).InitBias(frontMapNum, CUDA_Algo_Lib::g_idx_iter_init_bias);
			(*iter).InitErros(batch_size);
			(*iter).InitOutputMaps(batch_size);
			break;
		case 'O':
			(*iter).InitOutputKernel(frontMapNum, (*(iter - 1)).GetMapSize());
			(*iter).InitOutputLastStepDeltaKernel(frontMapNum, (*(iter - 1)).GetMapSize());//for adding momentum
			(*iter).InitBias(frontMapNum, CUDA_Algo_Lib::g_idx_iter_init_bias);
			(*iter).InitErros(batch_size);
			(*iter).InitOutputMaps(batch_size);
			break;
		default:
			break;
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::SetupTest(size_t batch_size)
{
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.begin();

	(*iter).InitOutputMaps(batch_size);
	iter++;
	for (iter; iter < vec_layers_.end(); iter++)
	{
		CUDA_Algo_Lib::g_idx_iter_init_bias = CUDA_Algo_Lib::g_idx_iter_init_bias + 1;

		size_t frontMapNum = (*(iter - 1)).GetOutMapNum();

		switch ((*iter).GetType())
		{
		case 'I':
			break;
		case 'C':
			// set map RectSize
			(*iter).SetMapSize((*(iter - 1)).GetMapSize().Substract((*iter).GetKernelSize(), 1));
			// initial convolution kernel_, quantities: frontMapNum*outMapNum_
			(*iter).InitKernel(frontMapNum);

			break;

		default:
			break;
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::BackPropagation(float* p_batch_data, float* p_batch_label)
{
	SetOutLayerErrors(p_batch_data, p_batch_label);
	SetHiddenLayerErrors();
}

void CUDA_Algo_Lib::CUDACNN::Forward(float* p_batch_data)
{
	SetInLayerOutput(p_batch_data);
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.begin()+1;
	//iter++;
	for (iter; iter < vec_layers_.end(); iter++)
	{
		switch ((*iter).GetType())
		{
		case 'C':
			SetConvOutput((*iter), (*(iter - 1)));
			break;
		case 'S':
			SetSampOutput((*iter), (*(iter - 1)));
			break;
		case 'H':
			SetFCHLayerOutput((*iter), (*(iter - 1)));
			break;
		case 'O':
			SetOutLayerOutput((*iter), (*(iter - 1)));
			break;
		default:
			break;
		}

	}
}

void CUDA_Algo_Lib::CUDACNN::SetInLayerOutput(float* p_batch_data)
{
	std::cout << "Execute CUDA_Algo_Lib::CUDACNN::SetInLayerOutput()" << std::endl;

	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.begin();

	RectSize map_size = (*iter).GetMapSize();
	size_t out_map_num = (*iter).GetOutMapNum();

	std::copy(p_batch_data, (p_batch_data + (batch_size_ * out_map_num * map_size.rows_ * map_size.cols_)), (*iter).vec_output_maps_.begin());
}
// for change the value in m_Layers
void CUDA_Algo_Lib::CUDACNN::SetConvOutput(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer)
{
	std::cout << "Execute CUDA_Algo_Lib::CUDACNN::SetConvOutput()" << std::endl;
	
	size_t layer_map_num = r_layer.GetOutMapNum();
	size_t lastlayer_map_num = r_lastlayer.GetOutMapNum();
	size_t lastlayer_map_x = r_lastlayer.GetMapSize().rows_;
	size_t lastlayer_map_y = r_lastlayer.GetMapSize().cols_;
	size_t layer_kernel_x = r_layer.GetKernelSize().rows_;
	size_t layer_kernel_y = r_layer.GetKernelSize().cols_;
	size_t layer_map_x = r_layer.GetMapSize().rows_;
	size_t layer_map_y = r_layer.GetMapSize().cols_;
	std::vector<float> vec_sum(layer_map_x * layer_map_y, 0.0);
	std::vector<float> vec_sum_now(layer_map_x * layer_map_y, 0.0);

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t i = 0; i < layer_map_num; i++)
		{
			for (size_t j = 0; j < lastlayer_map_num; j++)
			{
				size_t shift_idx_lastlayer_batch_map = idx_batch * lastlayer_map_num * lastlayer_map_x * lastlayer_map_y;
				size_t shift_idx_lastlayer_out_map = j * lastlayer_map_x * lastlayer_map_y;
				float* p_lastlayer_map = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;
				//float** lastMap;
				//lastMap = r_lastlayer.outputmaps_[idx_batch][j];				
				size_t shift_idx_layer_front_kernel = j * layer_map_num * layer_kernel_x * layer_kernel_y;
				size_t shift_idx_layer_out_kernel = i * layer_kernel_x * layer_kernel_y;
				float* p_layer_kernel = r_layer.vec_kernel_.data() + shift_idx_layer_front_kernel + shift_idx_layer_out_kernel;

				if (j == 0)
				{
					ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum.data());
					//each time we calculate one image of batch and also calculate relu 

				}
				else {
					ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum_now.data());
					CalConvArrayPlus(vec_sum_now.data(), vec_sum.data(), layer_map_x, layer_map_y);// sumNow 

				}
			}

			ActiveRelu(vec_sum.data(), r_layer.vec_bias_.at(i), layer_map_x, layer_map_y);//for relu active fun.

			size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
			size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
			float* p_layer_out_map = r_layer.vec_output_maps_.data() + shift_idx_layer_batch_map + shift_idx_layer_out_map;
			memcpy(p_layer_out_map, vec_sum.data(), (layer_map_x * layer_map_y * sizeof(float)));

		}
	}

}

void CUDA_Algo_Lib::CUDACNN::SetSampOutput(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer)
{
	std::cout << "Execute CUDA_Algo_Lib::CUDACNN::SetSampOutput()" << std::endl;

	size_t lastlayer_map_num = r_lastlayer.GetOutMapNum();
	size_t lastlayer_map_x = r_lastlayer.GetMapSize().rows_;
	size_t lastlayer_map_y = r_lastlayer.GetMapSize().cols_;
	size_t layer_map_x = r_layer.GetMapSize().rows_;
	size_t layer_map_y = r_layer.GetMapSize().cols_;
	RectSize scale_size = r_layer.GetScaleSize();
	std::vector<float> vec_samp_matrix(layer_map_x*layer_map_y, 0.0);

	float* p_lastlayer_map = NULL;
	size_t shift_idx_lastlayer_batch_map = 0;
	size_t shift_idx_lastlayer_out_map = 0;

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t i = 0; i < lastlayer_map_num; i++)
		{
			shift_idx_lastlayer_batch_map = idx_batch * lastlayer_map_num * lastlayer_map_x * lastlayer_map_y;
			shift_idx_lastlayer_out_map = i * lastlayer_map_x * lastlayer_map_y;
			p_lastlayer_map = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;			
			ScaleMatrix(p_lastlayer_map, scale_size, lastlayer_map_x, lastlayer_map_y, vec_samp_matrix.data());
			
			size_t shift_idx_layer_batch_map = idx_batch * lastlayer_map_num * layer_map_x * layer_map_y;
			size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
			float* p_layer_out_map = r_layer.vec_output_maps_.data() + shift_idx_layer_batch_map + shift_idx_layer_out_map;
			memcpy(p_layer_out_map, vec_samp_matrix.data(), (layer_map_x * layer_map_y * sizeof(float)));
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::SetFCHLayerOutput(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer)
{
	std::cout << "Execute CUDA_Algo_Lib::CUDACNN::SetFCHLayerOutput()" << std::endl;

	size_t layer_map_num = r_layer.GetOutMapNum();
	size_t lastlayer_map_num = r_lastlayer.GetOutMapNum();
	size_t lastlayer_map_x = r_lastlayer.GetMapSize().rows_;
	size_t lastlayer_map_y = r_lastlayer.GetMapSize().cols_;
	size_t layer_kernel_x = r_layer.GetKernelSize().rows_;
	size_t layer_kernel_y = r_layer.GetKernelSize().cols_;
	size_t layer_map_x = r_layer.GetMapSize().rows_;
	size_t layer_map_y = r_layer.GetMapSize().cols_;
	std::vector<float> vec_sum(layer_map_x * layer_map_y, 0.0);
	std::vector<float> vec_sum_now(layer_map_x * layer_map_y, 0.0);

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t i = 0; i < layer_map_num; i++)
		{
			for (size_t j = 0; j < lastlayer_map_num; j++)
			{
				size_t shift_idx_lastlayer_batch_map = idx_batch * lastlayer_map_num * lastlayer_map_x * lastlayer_map_y;
				size_t shift_idx_lastlayer_out_map = j * lastlayer_map_x * lastlayer_map_y;
				float* p_lastlayer_map = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;
			
				size_t shift_idx_layer_front_kernel = j * layer_map_num * layer_kernel_x * layer_kernel_y;
				size_t shift_idx_layer_out_kernel = i * layer_kernel_x * layer_kernel_y;
				float* p_layer_kernel = r_layer.vec_kernel_.data() + shift_idx_layer_front_kernel + shift_idx_layer_out_kernel;

				if (j == 0)
				{
					ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum.data());
					//each time we calculate one image of batch and also calculate relu 

				}
				else {
					ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum_now.data());
					CalFCHArrayPlus(vec_sum_now.data(), vec_sum.data(), layer_map_x, layer_map_y);// sumNow 

				}
			}

			//printf("ActiveRelu");
			ActiveRelu(vec_sum.data(), r_layer.vec_bias_.at(i), layer_map_x, layer_map_y);//for relu active fun.

			//SetValue(r_layer.outputmaps_[idx_batch][i], sum, layer_map_x, layer_map_y);
			size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
			size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
			float* p_layer_out_map = r_layer.vec_output_maps_.data() + shift_idx_layer_batch_map + shift_idx_layer_out_map;
			memcpy(p_layer_out_map, vec_sum.data(), (layer_map_x * layer_map_y * sizeof(float)));

		}

	}
}

// ReLU+Softmax function
void CUDA_Algo_Lib::CUDACNN::SetOutLayerOutput(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer)
{
	std::cout << "Execute CUDA_Algo_Lib::CUDACNN::SetOutLayerOutput()" << std::endl;

	size_t layer_map_num = r_layer.GetOutMapNum();
	size_t lastlayer_map_num = r_lastlayer.GetOutMapNum();
	size_t lastlayer_map_x = r_lastlayer.GetMapSize().rows_;
	size_t lastlayer_map_y = r_lastlayer.GetMapSize().cols_;
	size_t layer_kernel_x = r_layer.GetKernelSize().rows_;
	size_t layer_kernel_y = r_layer.GetKernelSize().cols_;
	size_t layer_map_x = r_layer.GetMapSize().rows_;
	size_t layer_map_y = r_layer.GetMapSize().cols_;
	std::vector<float> vec_sum(layer_map_x * layer_map_y, 0.0);
	std::vector<float> vec_sum_now(layer_map_x * layer_map_y, 0.0);
	std::vector<float> vec_sum_expone(batch_size_, 0.0);

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		//printf("ActiveRelu+softmax");
		//std::cout << "NO.of Batch: " << idx_batch << std::endl;
		for (size_t i = 0; i < layer_map_num; i++)
		{
			for (size_t j = 0; j < lastlayer_map_num; j++)
			{
				size_t shift_idx_lastlayer_batch_map = idx_batch * lastlayer_map_num * lastlayer_map_x * lastlayer_map_y;
				size_t shift_idx_lastlayer_out_map = j * lastlayer_map_x * lastlayer_map_y;
				float* p_lastlayer_map = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;
			
				size_t shift_idx_layer_front_kernel = j * layer_map_num * layer_kernel_x * layer_kernel_y;
				size_t shift_idx_layer_out_kernel = i * layer_kernel_x * layer_kernel_y;
				float* p_layer_kernel = r_layer.vec_kernel_.data() + shift_idx_layer_front_kernel + shift_idx_layer_out_kernel;

				if (j == 0)
				{
					ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum.data());
					//each time we calculate one image of batch and also calculate relu 

				}
				else {
					ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum_now.data());
					CalFCHArrayPlus(vec_sum_now.data(), vec_sum.data(), layer_map_x, layer_map_y);// sumNow 

				}
			}

			CalExpone(vec_sum.data(), r_layer.vec_bias_.at(i), layer_map_x, layer_map_y);

			size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
			size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
			float* p_layer_out_map = r_layer.vec_output_maps_.data() + shift_idx_layer_batch_map + shift_idx_layer_out_map;
			memcpy(p_layer_out_map, vec_sum.data(), (layer_map_x * layer_map_y * sizeof(float)));

		}

		for (size_t i = 0; i < layer_map_num; i++)
		{
			for (size_t ii = 0; ii < layer_map_x; ii++)
			{
				for (size_t jj = 0; jj < layer_map_y; jj++)
				{
					size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
					size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
					size_t shift_idx_layer_out_map_row = ii * layer_map_y;
					size_t idx_layer_out_map = shift_idx_layer_batch_map + shift_idx_layer_out_map + shift_idx_layer_out_map_row + jj;
					vec_sum_expone[idx_batch] += r_layer.vec_output_maps_.at(idx_layer_out_map);
				}
			}
		}

		for (size_t i = 0; i < layer_map_num; i++)
		{
			for (size_t ii = 0; ii < layer_map_x; ii++)
			{
				for (size_t jj = 0; jj < layer_map_y; jj++)
				{

					size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
					size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
					size_t shift_idx_layer_out_map_row = ii * layer_map_y;
					size_t idx_layer_out_map = shift_idx_layer_batch_map + shift_idx_layer_out_map + shift_idx_layer_out_map_row + jj;
					r_layer.vec_output_maps_[idx_layer_out_map] = r_layer.vec_output_maps_[idx_layer_out_map] / vec_sum_expone[idx_batch];
					
					std::cout << "Outputlayer's Softmax actual output(r_layer.outputmaps_[" << idx_batch << "][" << i << "][" << ii << "][" << jj << "]): " << r_layer.vec_output_maps_[idx_layer_out_map] << std::endl;
				}
			}
		}
	}

}

void CUDA_Algo_Lib::CUDACNN::SetOutLayerErrors(float* p_input_maps, float* p_target_labels)
{
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.end();
	iter--;
	size_t layer_outmap_num = (*iter).GetOutMapNum();
	float mean_error = 0.0, max_error = 0.0;

	//FILE* fy;
	//fy = fopen("./outputdata/error.txt", "a");

	////if( (err=fopen_s(&fy, "error.txt", "a")) != 0 )
	////	exit(1) ;

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t idx_map = 0; idx_map < layer_outmap_num; idx_map++)
		{
			//float val_out_map = (*iter).outputmaps_[idx_batch][idx_map][0][0];
			float val_target_label = p_target_labels[idx_batch * layer_outmap_num + idx_map];
			size_t shift_idx_layer_batch_map = idx_batch * layer_outmap_num * ((*iter).GetMapSize().rows_) * ((*iter).GetMapSize().cols_);
			size_t shift_idx_layer_out_map = idx_map * ((*iter).GetMapSize().rows_) * ((*iter).GetMapSize().cols_);
			size_t shift_idx_layer_out_map_row = 0 * ((*iter).GetMapSize().cols_);
			size_t idx_layer_out_map = shift_idx_layer_batch_map + shift_idx_layer_out_map + shift_idx_layer_out_map_row + 0;
			float val_out_map = (*iter).vec_output_maps_.at(idx_layer_out_map);

			//printf("Cross-entropy cost function for ReLU+Softmax");
			//Cross entropy for softmax form
			(*iter).SetError(idx_batch, idx_map, 0, 0, (val_target_label - val_out_map));
			mean_error = abs(val_target_label - val_out_map);

			//fprintf(fy, "%f ", mean_error);
			//// 			mean_error += abs(val_target_label-val_out_map);
			//// 			if (abs(val_target_label-val_out_map)>max_error)
			//// 			{
			//// 				max_error = abs(val_target_label-val_out_map);
			//// 			}
		}
		//fprintf(fy, "\n");
	}
	//fprintf(fy, "\n");
	//fclose(fy);
	//// 	std::cout<<"Mean error of each mini batch: "<<mean_error<<std::endl;
	//// 	std::cout<<"The max error of one output in mini batch: "<<max_error<<std::endl;
}

void CUDA_Algo_Lib::CUDACNN::SetFCHiddenLayerErrors(CUDA_Algo_Lib::CUDACNNLayer& Lastlayer, CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_nextlayer)//for add FC hiddenlayer
{
	size_t lastlayer_outmap_num = Lastlayer.GetOutMapNum();
	size_t layer_outmap_num = r_layer.GetOutMapNum();
	size_t layer_outmap_rows = r_layer.GetMapSize().rows_;
	size_t layer_outmap_cols = r_layer.GetMapSize().cols_;
	size_t nextlayer_outmap_num = r_nextlayer.GetOutMapNum();
	size_t nextlayer_outmap_rows = r_nextlayer.GetMapSize().rows_;
	size_t nextlayer_outmap_cols = r_nextlayer.GetMapSize().cols_;

	float* p_layer_outmap = nullptr;
	float* p_layer_error = nullptr;
	float* p_nextlayer_error = nullptr;
	std::vector<float> vec_layer_outmatrix;
	std::vector<float> vec_layer_outkroneckermatrix;
	vec_layer_outmatrix.reserve(layer_outmap_rows * layer_outmap_cols);
	vec_layer_outmatrix.resize(layer_outmap_rows * layer_outmap_cols);
	vec_layer_outkroneckermatrix.reserve(layer_outmap_rows * layer_outmap_cols);
	vec_layer_outkroneckermatrix.resize(layer_outmap_rows * layer_outmap_cols);
	RectSize layer_scale_size = r_layer.GetScaleSize();

	size_t nextlayer_kernel_rows = r_nextlayer.GetKernelSize().rows_;
	size_t nextlayer_kernel_cols = r_nextlayer.GetKernelSize().cols_;

	float* p_nextlayer_kernel = nullptr;

	std::vector<float> vec_derivative_active_fun;
	vec_derivative_active_fun.reserve(batch_size_ * layer_outmap_num);
	vec_derivative_active_fun.resize(batch_size_ * layer_outmap_num);

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
		{
			size_t shift_idx_layer_batch_map = idx_batch * layer_outmap_num * layer_outmap_rows * layer_outmap_cols;
			size_t shift_idx_layer_out_map = idx_layer_outmap * layer_outmap_rows * layer_outmap_cols;
			p_layer_outmap = r_layer.vec_output_maps_.data() + shift_idx_layer_batch_map + shift_idx_layer_out_map;
			
			r_layer.SetFCHLayerError(idx_batch, idx_layer_outmap, p_layer_outmap, layer_outmap_rows, layer_outmap_cols);

			//printf("derivative of ReLu");
			float* p_derivative_active_fun = vec_derivative_active_fun.data() + (idx_batch * layer_outmap_num + idx_layer_outmap);
			MatrixDreluFChidden(p_layer_outmap, layer_outmap_rows, layer_outmap_cols, p_derivative_active_fun);//for relu active fun.

		}
	}
	//printf("================================================================================\n");

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
		{
			p_layer_error = r_layer.GetError(idx_batch, idx_layer_outmap);
			p_layer_error[0 * layer_outmap_num + 0] = vec_derivative_active_fun.at(idx_batch * layer_outmap_num + idx_layer_outmap);

		}
	}
	//printf("================================================================================\n");

	std::vector<float> vec_sum_local_gradient((batch_size_ * layer_outmap_num), 0.0);

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
		{
			for (size_t idx_nextlayer_outmap = 0; idx_nextlayer_outmap < nextlayer_outmap_num; idx_nextlayer_outmap++)
			{
				p_layer_error = r_nextlayer.GetError(idx_batch, idx_nextlayer_outmap);
				p_nextlayer_kernel = r_nextlayer.GetKernel(idx_layer_outmap, idx_nextlayer_outmap);

				vec_sum_local_gradient[idx_batch * layer_outmap_num + idx_layer_outmap] += p_layer_error[0 * nextlayer_outmap_num + 0] * p_nextlayer_kernel[0 * nextlayer_kernel_cols + 0];

			}

		}
	}

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
		{
			p_layer_error = r_layer.GetError(idx_batch, idx_layer_outmap);
			if (0.0 == p_layer_error[0 * nextlayer_outmap_num + 0])
			{
				p_layer_error[0 * nextlayer_outmap_num + 0] = p_layer_error[0 * nextlayer_outmap_num + 0] * vec_sum_local_gradient[idx_batch * layer_outmap_num + idx_layer_outmap];

				p_layer_error[0 * nextlayer_outmap_num + 0] = abs(p_layer_error[0 * nextlayer_outmap_num + 0]);
			}
			else {
				p_layer_error[0 * nextlayer_outmap_num + 0] = p_layer_error[0 * nextlayer_outmap_num + 0] * vec_sum_local_gradient[idx_batch * layer_outmap_num + idx_layer_outmap];
			}

			r_layer.SetFCHLayerError(idx_batch, idx_layer_outmap, p_layer_error, 0, 0);

		}
	}

}

void CUDA_Algo_Lib::CUDACNN::SetHiddenLayerErrors()
{
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.end();
	iter = iter - 2;
	for (iter; iter > vec_layers_.begin(); iter--)
	{
		switch ((*(iter)).GetType())
		{
		case 'C':
			SetConvErrors((*iter), (*(iter + 1)));
			break;
		case 'S':
			SetSampErrors((*iter), (*(iter + 1)));
			break;
		case 'H':
			SetFCHiddenLayerErrors((*(iter - 1)), (*iter), (*(iter + 1)));
			break;
		default:
			break;
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::SetSampErrors(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_nextlayer)
{
	size_t layer_outmap_num = r_layer.GetOutMapNum();
	size_t layer_outmap_rows = r_layer.GetMapSize().rows_;
	size_t layer_outmap_cols = r_layer.GetMapSize().cols_;
	size_t nextlayer_outmap_num = r_nextlayer.GetOutMapNum();
	size_t nextlayer_outmap_rows = r_nextlayer.GetMapSize().rows_;
	size_t nextlayer_outmap_cols = r_nextlayer.GetMapSize().cols_;
	size_t nextlayer_kernel_rows = r_nextlayer.GetKernelSize().rows_;
	size_t nextlayer_kernel_cols = r_nextlayer.GetKernelSize().cols_;

	float* p_nextlayer_error = nullptr;
	float* p_nextlayer_kernel = nullptr;
	std::vector<float> vec_sum(layer_outmap_rows * layer_outmap_cols, 0.0);
	std::vector<float> vec_sum_now(layer_outmap_rows * layer_outmap_cols, 0.0);
	std::vector<float> vec_rot_matrix(nextlayer_kernel_rows * nextlayer_kernel_cols, 0.0);
	std::vector<float> vec_nextlayer_extend_matrix((nextlayer_outmap_rows+2*(nextlayer_kernel_rows-1)) * (nextlayer_outmap_cols+2*(nextlayer_kernel_cols-1)), 0.0);

	//calculate
	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
		{
			for (size_t idx_nextlayer_outmap = 0; idx_nextlayer_outmap < nextlayer_outmap_num; idx_nextlayer_outmap++)
			{

				p_nextlayer_error = r_nextlayer.GetError(idx_batch, idx_nextlayer_outmap);
				p_nextlayer_kernel = r_nextlayer.GetKernel(idx_layer_outmap, idx_nextlayer_outmap);
				if (idx_nextlayer_outmap == 0)
				{
					Rot180(p_nextlayer_kernel, nextlayer_kernel_rows, nextlayer_kernel_cols, vec_rot_matrix.data());
					ConvNSampFull(p_nextlayer_error, vec_rot_matrix.data(), nextlayer_outmap_rows, nextlayer_outmap_cols, nextlayer_kernel_rows, nextlayer_kernel_cols, vec_sum.data(), vec_nextlayer_extend_matrix.data());

				}
				else
				{
					Rot180(p_nextlayer_kernel, nextlayer_kernel_rows, nextlayer_kernel_cols, vec_rot_matrix.data());
					ConvNSampFull(p_nextlayer_error, vec_rot_matrix.data(), nextlayer_outmap_rows, nextlayer_outmap_cols, nextlayer_kernel_rows, nextlayer_kernel_cols, vec_sum_now.data(), vec_nextlayer_extend_matrix.data());
					CalSampArrayPlus(vec_sum_now.data(), vec_sum.data(), layer_outmap_rows, layer_outmap_cols);

				}

			}
			r_layer.SetSampLayerError(idx_batch, idx_layer_outmap, vec_sum.data(), layer_outmap_rows, layer_outmap_cols);
		}
	}

}

void CUDA_Algo_Lib::CUDACNN::SetConvErrors(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_nextlayer)
{
	size_t layer_outmap_num = r_layer.GetOutMapNum();
	size_t layer_outmap_rows = r_layer.GetMapSize().rows_;
	size_t layer_outmap_cols = r_layer.GetMapSize().cols_;
	size_t nextlayer_outmap_rows = r_nextlayer.GetMapSize().rows_;
	size_t nextlayer_outmap_cols = r_nextlayer.GetMapSize().cols_;

	float* p_nextlayer_error = nullptr;
	float* p_layer_outmap = nullptr;
	std::vector<float> vec_layer_outmatrix;
	std::vector<float> vec_layer_outkroneckermatrix;
	vec_layer_outmatrix.reserve(layer_outmap_rows * layer_outmap_cols);
	vec_layer_outkroneckermatrix.reserve(layer_outmap_rows * layer_outmap_cols);
	vec_layer_outmatrix.resize(layer_outmap_rows * layer_outmap_cols);
	vec_layer_outkroneckermatrix.resize(layer_outmap_rows * layer_outmap_cols);
	RectSize layer_scale_size = r_layer.GetScaleSize();

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
		{
			layer_scale_size = r_nextlayer.GetScaleSize();
			p_nextlayer_error = r_nextlayer.GetError(idx_batch, idx_layer_outmap);
			size_t shift_idx_layer_batch_map = idx_batch * layer_outmap_num * layer_outmap_rows * layer_outmap_cols;
			size_t shift_idx_layer_out_map = idx_layer_outmap * layer_outmap_rows * layer_outmap_cols;
			p_layer_outmap = r_layer.vec_output_maps_.data() + shift_idx_layer_batch_map + shift_idx_layer_out_map;

			//printf("derivative of ReLu");
			//derivative of ReLu
			MatrixDreluConv(p_layer_outmap, layer_outmap_rows, layer_outmap_cols, vec_layer_outmatrix.data());//for relu active fun.

			CalKronecker(p_nextlayer_error, layer_scale_size, nextlayer_outmap_rows, nextlayer_outmap_cols, vec_layer_outkroneckermatrix.data(), layer_outmap_rows, layer_outmap_cols);
			CalMatrixMultiply(vec_layer_outmatrix.data(), vec_layer_outkroneckermatrix.data(), layer_outmap_rows, layer_outmap_cols);

			r_layer.SetConvLayerError(idx_batch, idx_layer_outmap, vec_layer_outmatrix.data(), layer_outmap_rows, layer_outmap_cols);

		}
	}
}

void CUDA_Algo_Lib::CUDACNN::UpdateParas()
{
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.begin();
	iter++;

	CUDA_Algo_Lib::g_idx_itor = 0;//begining at index 0 r_layer
	char str_file_kernel[1000];// initialized properly
	char str_file_bias[1000];// initialized properly

	for (iter; iter < vec_layers_.end(); iter++)
	{
		CUDA_Algo_Lib::g_idx_itor = CUDA_Algo_Lib::g_idx_itor + 1;
		sprintf(str_file_kernel, "./data/kernel_weight/kernel_weight_%d_%d", CUDA_Algo_Lib::g_idx_itor, (*iter).GetType());
		sprintf(str_file_bias, "./data/bias/bias_%d_%d", CUDA_Algo_Lib::g_idx_itor, (*iter).GetType());
		//printf("%s", str_file_kernel);

		switch ((*iter).GetType())
		{
		case 'C':
			UpdateKernels(*iter, *(iter - 1), str_file_kernel, eta_conv_, alpha_conv_);
			UpdateBias(*iter, str_file_bias, eta_conv_);
			break;
		case 'H':
			UpdateKernels(*iter, *(iter - 1), str_file_kernel, eta_fc_, alpha_fc_);
			UpdateBias(*iter, str_file_bias, eta_fc_);
			break;
		case 'O':
			UpdateKernels(*iter, *(iter - 1), str_file_kernel, eta_fc_, alpha_fc_);
			UpdateBias(*iter, str_file_bias, eta_fc_);
			break;
		default:
			break;
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::UpdateBias(CUDA_Algo_Lib::CUDACNNLayer& r_layer, char* str_File_Bias, float eta)
{
	size_t layer_outmap_num = r_layer.GetOutMapNum();
	size_t layer_outmap_rows = r_layer.GetMapSize().rows_;
	size_t layer_outmap_cols = r_layer.GetMapSize().cols_;
	float* p_layer_error = r_layer.vec_errors_.data();
	std::vector<float> vec_error((layer_outmap_rows* layer_outmap_cols), 0.0);
	float deltaBias = 0.0;

	for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
	{

		CalErrorsSum(p_layer_error, idx_layer_outmap, layer_outmap_num, layer_outmap_rows, layer_outmap_cols, batch_size_, vec_error.data());
		deltaBias = (CalErrorSum(vec_error.data(), layer_outmap_rows, layer_outmap_cols) / ((float)batch_size_));
		r_layer.vec_bias_.at(idx_layer_outmap) += (eta * deltaBias);

		/***save bias_***/
		if ((CUDA_Algo_Lib::g_iteration_num - 1) == CUDA_Algo_Lib::g_idx_iteration_num) {
			char str_file_bias_1[1000];
			sprintf(str_file_bias_1, "%s_%d.txt", str_File_Bias, idx_layer_outmap);
			FILE* fp_bias = fopen(str_file_bias_1, "w");

			fprintf(fp_bias, "%f ", r_layer.vec_bias_.at(idx_layer_outmap));
			fprintf(fp_bias, "\n");

			fclose(fp_bias);
		}
	}

}

void CUDA_Algo_Lib::CUDACNN::UpdateKernels(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer, char* str_File_Kernel, float eta, float alpha)
{
	size_t lastlayer_outmap_num = r_lastlayer.GetOutMapNum();
	size_t lastlayer_outmap_rows = r_lastlayer.GetMapSize().rows_;
	size_t lastlayer_outmap_cols = r_lastlayer.GetMapSize().cols_;
	size_t layer_outmap_num = r_layer.GetOutMapNum();
	size_t layer_outmap_rows = r_layer.GetMapSize().rows_;
	size_t layer_outmap_cols = r_layer.GetMapSize().cols_;
	size_t layer_kernel_rows = r_layer.GetKernelSize().rows_;
	size_t layer_kernel_cols = r_layer.GetKernelSize().cols_;


	std::vector<float> vec_delta_kernel_1((layer_kernel_rows * layer_kernel_cols), 0.0);
	std::vector<float> vec_delta_kernel_2((layer_kernel_rows * layer_kernel_cols), 0.0);
	std::vector<float> vec_delta_now((layer_kernel_rows * layer_kernel_cols), 0.0);
	float* p_layer_error = nullptr;
	float* p_lastlayer_outmap = nullptr;
	float* p_layer_laststep_delta_kernel = nullptr;
	float* p_layer_kernel = nullptr;

	for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
	{
		for (size_t idx_lastlayer_outmap = 0; idx_lastlayer_outmap < lastlayer_outmap_num; idx_lastlayer_outmap++)
		{
			for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
			{
				p_layer_error = r_layer.GetError(idx_batch, idx_layer_outmap);
				if (idx_batch == 0) {
					size_t shift_idx_lastlayer_batch_map = idx_batch * lastlayer_outmap_num * lastlayer_outmap_rows * lastlayer_outmap_cols;
					size_t shift_idx_lastlayer_out_map = idx_lastlayer_outmap * lastlayer_outmap_rows * lastlayer_outmap_cols;
					p_lastlayer_outmap = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;
					ConvNValid(p_lastlayer_outmap, p_layer_error, lastlayer_outmap_rows, lastlayer_outmap_cols, layer_outmap_rows, layer_outmap_cols, vec_delta_kernel_1.data());
				}
				else {
					size_t shift_idx_lastlayer_batch_map = idx_batch * lastlayer_outmap_num * lastlayer_outmap_rows * lastlayer_outmap_cols;
					size_t shift_idx_lastlayer_out_map = idx_lastlayer_outmap * lastlayer_outmap_rows * lastlayer_outmap_cols;
					p_lastlayer_outmap = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;
					ConvNValid(p_lastlayer_outmap, p_layer_error, lastlayer_outmap_rows, lastlayer_outmap_cols, layer_outmap_rows, layer_outmap_cols, vec_delta_now.data());
					CalConvArrayPlus(vec_delta_now.data(), vec_delta_kernel_1.data(), layer_kernel_rows, layer_kernel_cols);
				}
			}
			size_t shift_idx_layer_kernel_lastlayer = idx_lastlayer_outmap * layer_outmap_num * layer_kernel_rows * layer_kernel_cols;
			size_t shift_idx_layer_kernel_layer = idx_layer_outmap * layer_kernel_rows * layer_kernel_cols;
			p_layer_laststep_delta_kernel = r_layer.vec_laststep_delta_kernel_.data() + shift_idx_layer_kernel_lastlayer + shift_idx_layer_kernel_layer;
			p_layer_kernel = r_layer.vec_kernel_.data() + shift_idx_layer_kernel_lastlayer + shift_idx_layer_kernel_layer;
			SetKernelValue(vec_delta_kernel_2.data(), p_layer_laststep_delta_kernel, layer_kernel_rows, layer_kernel_cols);
			CalArrayMultiply(vec_delta_kernel_2.data(), alpha, layer_kernel_rows, layer_kernel_cols);//for adding momentum
			CalArrayPlus(vec_delta_kernel_2.data(), p_layer_kernel, layer_kernel_rows, layer_kernel_cols);//for adding momentum
			CalArrayDivide(vec_delta_kernel_1.data(), batch_size_, layer_kernel_rows, layer_kernel_cols);
			CalArrayMultiply(vec_delta_kernel_1.data(), eta, layer_kernel_rows, layer_kernel_cols);
			SetKernelValue(p_layer_laststep_delta_kernel, vec_delta_kernel_1.data(), layer_kernel_rows, layer_kernel_cols);//for adding momentum
			CalArrayPlus(vec_delta_kernel_1.data(), p_layer_kernel, layer_kernel_rows, layer_kernel_cols);

			/***save kernel_ weight***/
			if ((CUDA_Algo_Lib::g_iteration_num - 1) == CUDA_Algo_Lib::g_idx_iteration_num) {
				char str_file_kernel_1[1000];
				sprintf(str_file_kernel_1, "%s_%d_%d.txt", str_File_Kernel, idx_lastlayer_outmap, idx_layer_outmap);

				FILE* fp = fopen(str_file_kernel_1, "w");
				size_t shift_idx_layer_kernel_lastlayer = 0;
				size_t shift_idx_layer_kernel_layer = 0;
				size_t idx_layer_kernel = 0;

				for (size_t mm = 0; mm < layer_kernel_rows; mm++)
				{
					for (size_t nn = 0; nn < layer_kernel_cols; nn++)
					{
						shift_idx_layer_kernel_lastlayer = idx_lastlayer_outmap * layer_outmap_num * layer_kernel_rows * layer_kernel_cols;
						shift_idx_layer_kernel_layer = idx_layer_outmap * layer_kernel_rows * layer_kernel_cols;
						idx_layer_kernel = shift_idx_layer_kernel_lastlayer + shift_idx_layer_kernel_layer + (mm * layer_kernel_cols + nn);

						fprintf(fp, "%f ", r_layer.vec_kernel_.at(idx_layer_kernel));
					}

				}
				fprintf(fp, "\n");
				fclose(fp);
			}

		}
	}

}

void CUDA_Algo_Lib::CUDACNN::LoadParas()
{
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.begin();
	iter++;

	CUDA_Algo_Lib::g_idx_itor = 0;//begining at index 0 r_layer
	char str_file_kernel[1000];// initialized properly
	char str_file_bias[1000];// initialized properly

	for (iter; iter < vec_layers_.end(); iter++)
	{
		CUDA_Algo_Lib::g_idx_itor = CUDA_Algo_Lib::g_idx_itor + 1;
		sprintf(str_file_kernel, "./data/kernel_weight/kernel_weight_%d_%d", CUDA_Algo_Lib::g_idx_itor, (*iter).GetType());
		sprintf(str_file_bias, "./data/bias/bias_%d_%d", CUDA_Algo_Lib::g_idx_itor, (*iter).GetType());
		//printf("%s", str_file_kernel);

		switch ((*iter).GetType())
		{
		case 'C':
			LoadKernels(*iter, *(iter - 1), str_file_kernel);
			LoadBias(*iter, str_file_bias);
			break;
		case 'H':
			LoadKernels(*iter, *(iter - 1), str_file_kernel);
			LoadBias(*iter, str_file_bias);
			break;
		case 'O':
			LoadKernels(*iter, *(iter - 1), str_file_kernel);
			LoadBias(*iter, str_file_bias);
			break;
		default:
			break;
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::LoadBias(CUDA_Algo_Lib::CUDACNNLayer& r_layer, char* str_File_Bias)
{
	size_t layer_outmap_num = r_layer.GetOutMapNum();
	float bias = 0.0;

	for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
	{
		bias = 0.0;
		/***load bias***/
		char str_file_bias_1[1000];
		sprintf(str_file_bias_1, "%s_%d.txt", str_File_Bias, idx_layer_outmap);
		printf("%s\n", str_file_bias_1);
		FILE* fp_bias = fopen(str_file_bias_1, "r");
		fscanf(fp_bias, "%f ", &bias);
		fclose(fp_bias);

		r_layer.vec_bias_.at(idx_layer_outmap) = bias;
		printf("bias: %f\n", r_layer.vec_bias_.at(idx_layer_outmap));
	}
	
}

void CUDA_Algo_Lib::CUDACNN::LoadKernels(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer, char* str_File_Kernel)
{
	
	const size_t lastlayer_outmap_num = r_lastlayer.GetOutMapNum();
	const size_t lastlayer_outmap_rows = r_lastlayer.GetMapSize().rows_;
	const size_t lastlayer_outmap_cols = r_lastlayer.GetMapSize().cols_;
	const size_t layer_outmap_num = r_layer.GetOutMapNum();
	const size_t layer_outmap_rows = r_layer.GetMapSize().rows_;
	const size_t layer_outmap_cols = r_layer.GetMapSize().cols_;
	const size_t layer_kernel_rows = r_layer.GetKernelSize().rows_;
	const size_t layer_kernel_cols = r_layer.GetKernelSize().cols_;

	size_t shift_idx_layer_kernel_lastlayer = 0;
	size_t shift_idx_layer_kernel_layer = 0;
	size_t idx_layer_kernel = 0;

	std::vector<float> vec_kernel((layer_kernel_rows * layer_kernel_cols), 0.0);

	for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
	{
		for (size_t idx_lastlayer_outmap = 0; idx_lastlayer_outmap < lastlayer_outmap_num; idx_lastlayer_outmap++)
		{
			/***load kernel_ weight***/
			char str_file_kernel_1[1000];
			sprintf(str_file_kernel_1, "%s_%d_%d.txt", str_File_Kernel, idx_lastlayer_outmap, idx_layer_outmap);
			printf("%s\n", str_file_kernel_1);
			FILE* fp_kernel = fopen(str_file_kernel_1, "r");

			for (size_t mm = 0; mm < layer_kernel_rows; mm++)
			{
				for (size_t nn = 0; nn < layer_kernel_cols; nn++)
				{
					shift_idx_layer_kernel_lastlayer = idx_lastlayer_outmap * layer_outmap_num * layer_kernel_rows * layer_kernel_cols;
					shift_idx_layer_kernel_layer = idx_layer_outmap * layer_kernel_rows * layer_kernel_cols;
					idx_layer_kernel = shift_idx_layer_kernel_lastlayer + shift_idx_layer_kernel_layer + (mm * layer_kernel_cols + nn);

					fscanf(fp_kernel, "%f ", (vec_kernel.data()+(mm * layer_kernel_cols + nn)));
					r_layer.vec_kernel_.at(idx_layer_kernel) = vec_kernel.at(mm * layer_kernel_cols + nn);
					printf("kernel_: %f\n", r_layer.vec_kernel_.at(idx_layer_kernel));
				}

			}
			fclose(fp_kernel);
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::Inference(CUDA_Algo_Lib::DatasetLoadingParamPKG& r_dataset_param)
{
	std::cout << "Start Inference" << std::endl;

	size_t total_false = 0, false_1 = 0, false_2 = 0, predict, real;
	size_t total_num_iter = r_dataset_param.total_num_images_ / batch_size_;

	float* p_inference_batch_data = nullptr;
	float* p_inference_batch_label = nullptr;
	std::vector<float> vec_inference_batch_data;
	std::vector<float> vec_inference_batch_label;
	vec_inference_batch_data.reserve(batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_);
	vec_inference_batch_data.resize(batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_);
	vec_inference_batch_label.reserve(batch_size_ * r_dataset_param.num_output_cls_);
	vec_inference_batch_label.resize(batch_size_ * r_dataset_param.num_output_cls_);

	FILE* p_file_error_predict_neg = fopen("./outputdata/error_predict_neg_filename.txt", "w");
	FILE* p_file_error_predict_pos = fopen("./outputdata/error_predict_pos_filename.txt", "w");
	for (size_t idx_iteration = 0; idx_iteration < total_num_iter; idx_iteration++)
	{
		std::cout << "NO.of iteration(testing): " << idx_iteration << std::endl;
		size_t idx_inference_dataset_batch = idx_iteration % (r_dataset_param.total_num_images_ / batch_size_);
		for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
		{
			std::cout << "NO.of batch(testing): " << idx_batch << std::endl;
		
			std::vector<float>::iterator shift_begin_iter_loaded_dataset_batch_data;
			std::vector<float>::iterator shift_begin_iter_loaded_dataset_batch_label;
			shift_begin_iter_loaded_dataset_batch_data = r_dataset_param.vec_images_.begin() + (idx_inference_dataset_batch * batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_);
			shift_begin_iter_loaded_dataset_batch_label = r_dataset_param.vec_labels_.begin() + (idx_inference_dataset_batch * batch_size_ * r_dataset_param.num_output_cls_);
			vec_inference_batch_data.assign(shift_begin_iter_loaded_dataset_batch_data, (shift_begin_iter_loaded_dataset_batch_data + (batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_)));
			vec_inference_batch_label.assign(shift_begin_iter_loaded_dataset_batch_label, (shift_begin_iter_loaded_dataset_batch_label + (batch_size_ * r_dataset_param.num_output_cls_)));

		}

		Forward(vec_inference_batch_data.data());
		CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.end();
		iter--;
		for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
		{
			std::cout << idx_batch << std::endl;

			size_t layer_outmap_num = (*iter).GetOutMapNum();
			size_t layer_outmap_rows = (*iter).GetMapSize().rows_;
			size_t layer_outmap_cols = (*iter).GetMapSize().cols_;
			size_t shift_idx_layer_batch_map = idx_batch * layer_outmap_num * layer_outmap_rows * layer_outmap_cols;
			float* p_layer_batchmap = (*iter).vec_output_maps_.data() + shift_idx_layer_batch_map;
			predict = FindIndex(p_layer_batchmap, layer_outmap_num, layer_outmap_rows, layer_outmap_cols);

			float* p_batch_gt_label = vec_inference_batch_label.data() + (idx_batch * r_dataset_param.num_output_cls_);
			real = FindIndex(p_batch_gt_label, r_dataset_param.num_output_cls_);


			//predict For batch size=2
			if (0 == idx_batch) {
				if (predict != real)
				{
					false_1++;
					//num_charaters_neg1 = sprintf(_input_negfilename, "%s%d%s", _negfilepath, idx_iteration, _imgfileextension);
					//printf("error predict-number of charaters: %d, string: \"%s\"\n", num_charaters_neg1, _input_negfilename);
					//fprintf(p_file_error_predict_neg, "%s\n", _input_negfilename);

				}
			}
			else if (1 == idx_batch) {
				if (predict != real)
				{
					false_2++;
					//num_charaters_pos1 = sprintf(_input_posfilename, "%s%d%s", _posfilepath, idx_iteration, _imgfileextension);
					//num_charaters_pos1 = sprintf(_input_posfilename, "%s%d", _posfilepath, idx_iteration);
					//printf("error predict-number of charaters: %d, string: \"%s\"\n", num_charaters_pos1, _input_posfilename);
					//fprintf(p_file_error_predict_pos, "%s\n", _input_posfilename);
				}
			}
		}
	}

	total_false = false_1 + false_2;

	std::cout << "+++++++Finish Inference+++++++" << std::endl;
	std::cout << "Error predict number of neg: " << false_1 << std::endl;
	std::cout << "Error rate of neg: " << (float)false_1 / (float)r_dataset_param.num_neg_images_ << std::endl;
	std::cout << "Error predict number of pos: " << false_2 << std::endl;
	std::cout << "Error rate of pos: " << (float)false_2 / (float)r_dataset_param.num_pos_images_ << std::endl;
	std::cout << "Error predict total number: " << total_false << std::endl;
	std::cout << "Total error rate: " << (float)total_false / (float)r_dataset_param.total_num_images_ << std::endl << std::endl;

	FILE* p_file_false_metrics;
	p_file_false_metrics = fopen("./outputdata/false_metrics.txt", "a");
	/*
	if( (err=fopen_s(&p_file_false_metrics, "fausePrun.txt", "a")) != 0 )
		exit(1) ;
	*/
	CUDA_Algo_Lib::g_idx_epoch++;
	fprintf(p_file_false_metrics, "epoch: %4d\n", CUDA_Algo_Lib::g_idx_epoch);
	fprintf(p_file_false_metrics, "neg: %4d %8f\n", false_1, (float)false_1 / (float)r_dataset_param.num_neg_images_);
	fprintf(p_file_false_metrics, "pos: %4d %8f\n", false_2, (float)false_2 / (float)r_dataset_param.num_pos_images_);
	fprintf(p_file_false_metrics, "total: %4d %8f\n\n", total_false, (float)total_false / (float)r_dataset_param.total_num_images_);
	fclose(p_file_false_metrics);
	fclose(p_file_error_predict_pos);
	fclose(p_file_error_predict_neg);

}

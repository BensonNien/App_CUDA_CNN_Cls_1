#include "hip/hip_runtime.h"
/******************************************************************************
Date:  2022/09
Author: CHU-MIN, NIEN
Description: CUDA ver.
******************************************************************************/
#include <algorithm>

#include "CUDACNNCls.cuh"
#include "CUDACNNLayer.cuh"

// CUDA_Algo_Lib::CUDACNN

#define DERIV_ACTIVE_RELU(S) 1 // derivative of the relu as a function of the relu's output
namespace CUDA_Algo_Lib
{
	size_t g_idx_epoch = 0;//index of epoch
	size_t g_idx_itor = 0;//index of iterator
	size_t g_idx_iter_init_bias = 0;//index of iterator for initialize bias
	size_t g_idx_iteration_num = 0;//index of iteration
	size_t g_iteration_num = 0;//number of g_iteration_num
}

hipError_t CUDA_Algo_Lib::CUDACNN::InitCUDADevice()
{
	printf("========= CUDA_Algo_Lib::CUDACNN::InitCUDADevice() Start =========\n");
	// part1, check the number of device
	int  iDeviceCount = 0;
	hipGetDeviceCount(&iDeviceCount);
	printf("Number of GPU: %d\n", iDeviceCount);

	if (iDeviceCount == 0)
	{
		printf("No supported GPU!\n");
	}

	// part2, output information of each device
	for (int i = 0; i < iDeviceCount; ++i)
	{
		printf("=== GPU Device ID: %i ===\n", i);
		hipDeviceProp_t  sDeviceProp;
		hipGetDeviceProperties(&sDeviceProp, i);
		printf("Device name: %s\n", sDeviceProp.name);
		printf("Device memory: %lld\n", sDeviceProp.totalGlobalMem);
		printf("Memory per-block: %lld\n", sDeviceProp.sharedMemPerBlock);
		printf("Register per-block: %lld\n", sDeviceProp.regsPerBlock);
		printf("Warp size: %lld\n", sDeviceProp.warpSize);
		printf("Memory pitch: %lld\n", sDeviceProp.memPitch);
		printf("Constant Memory: %lld\n", sDeviceProp.totalConstMem);
		printf("Max thread per-block: %lld\n", sDeviceProp.maxThreadsPerBlock);
		printf("Max thread dim: ( %lld, %lld, %lld )\n", sDeviceProp.maxThreadsDim[0], sDeviceProp.maxThreadsDim[1], sDeviceProp.maxThreadsDim[2]);
		printf("Max grid size: ( %lld, %lld, %lld )\n", sDeviceProp.maxGridSize[0], sDeviceProp.maxGridSize[1], sDeviceProp.maxGridSize[2]);
		printf("Ver: %lld.%lld\n", sDeviceProp.major, sDeviceProp.minor);
		printf("Clock: %lld\n", sDeviceProp.clockRate);
		printf("textureAlignment: %lld\n", sDeviceProp.textureAlignment);
	}

	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}
	printf("========= CUDA_Algo_Lib::CUDACNN::InitCUDADevice() End =========\n");
	return cudaStatus;
}

void CUDA_Algo_Lib::CUDACNN::Train(CUDA_Algo_Lib::DatasetLoadingParamPKG& r_dataset_param)
{
	std::cout << "Start train" << std::endl;

	CUDA_Algo_Lib::g_iteration_num = r_dataset_param.total_num_images_ / batch_size_;
	if ((r_dataset_param.total_num_images_ % batch_size_) != 0)
	{
		std::cout << "Please reset CUDA_Algo_Lib::CUDACNN::batch_size_!" << std::endl;
	}

	float* p_train_batch_data = nullptr;
	float* p_train_batch_label = nullptr;
	std::vector<float> vec_train_batch_data;
	std::vector<float> vec_train_batch_label;
	vec_train_batch_data.reserve(batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_);
	vec_train_batch_data.resize(batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_);
	vec_train_batch_label.reserve(batch_size_ * r_dataset_param.num_output_cls_);
	vec_train_batch_label.resize(batch_size_ * r_dataset_param.num_output_cls_);

	for (CUDA_Algo_Lib::g_idx_iteration_num = 0; CUDA_Algo_Lib::g_idx_iteration_num < CUDA_Algo_Lib::g_iteration_num; CUDA_Algo_Lib::g_idx_iteration_num++)
	{
		std::cout << "NO.of iteration(training): " << CUDA_Algo_Lib::g_idx_iteration_num << std::endl;
		size_t idx_loaded_dataset_batch = CUDA_Algo_Lib::g_idx_iteration_num % (r_dataset_param.total_num_images_ / batch_size_);
		for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
		{
			std::cout << "NO.of batch(training): " << idx_batch << std::endl;

			std::vector<float>::iterator shift_begin_iter_loaded_dataset_batch_data;
			std::vector<float>::iterator shift_begin_iter_loaded_dataset_batch_label;
			shift_begin_iter_loaded_dataset_batch_data = r_dataset_param.vec_images_.begin() + (idx_loaded_dataset_batch * batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_);
			shift_begin_iter_loaded_dataset_batch_label = r_dataset_param.vec_labels_.begin() + (idx_loaded_dataset_batch * batch_size_ * r_dataset_param.num_output_cls_);
			vec_train_batch_data.assign(shift_begin_iter_loaded_dataset_batch_data, (shift_begin_iter_loaded_dataset_batch_data + (batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_)));
			vec_train_batch_label.assign(shift_begin_iter_loaded_dataset_batch_label, (shift_begin_iter_loaded_dataset_batch_label + (batch_size_ * r_dataset_param.num_output_cls_)));

		}


		Forward(vec_train_batch_data.data());
		BackPropagation(vec_train_batch_data.data(), vec_train_batch_label.data());
		UpdateParas();


	}
	std::cout << "Finish train" << std::endl;

}

void CUDA_Algo_Lib::CUDACNN::Setup(size_t batch_size)
{
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.begin();

	(*iter).InitOutputMaps(batch_size);
	iter++;
	for (iter; iter < vec_layers_.end(); iter++)
	{
		CUDA_Algo_Lib::g_idx_iter_init_bias = CUDA_Algo_Lib::g_idx_iter_init_bias + 1;

		size_t frontMapNum = (*(iter - 1)).GetOutMapNum();

		switch ((*iter).GetType())
		{
		case 'I':
			break;
		case 'C':
			// set map RectSize
			(*iter).SetMapSize((*(iter - 1)).GetMapSize().Substract((*iter).GetKernelSize(), 1));
			// initial convolution kernel_, quantities: frontMapNum*outMapNum_
			(*iter).InitKernel(frontMapNum);
			(*iter).InitLastStepDeltaKernel(frontMapNum);//for adding momentum
			//each map has one bias_, so frontMapNum is not necessary
			(*iter).InitBias(frontMapNum, CUDA_Algo_Lib::g_idx_iter_init_bias);
			(*iter).InitErros(batch_size);
			// each r_layer should initialize output map
			(*iter).InitOutputMaps(batch_size);
			break;
		case 'S':
			(*iter).SetOutMapNum((frontMapNum));
			(*iter).SetMapSize((*(iter - 1)).GetMapSize().Divide((*iter).GetScaleSize()));
			(*iter).InitErros(batch_size);
			(*iter).InitOutputMaps(batch_size);
			break;
		case 'H':
			(*iter).InitOutputKernel(frontMapNum, (*(iter - 1)).GetMapSize());
			(*iter).InitOutputLastStepDeltaKernel(frontMapNum, (*(iter - 1)).GetMapSize());//for adding momentum			
			(*iter).InitBias(frontMapNum, CUDA_Algo_Lib::g_idx_iter_init_bias);
			(*iter).InitErros(batch_size);
			(*iter).InitOutputMaps(batch_size);
			break;
		case 'O':
			(*iter).InitOutputKernel(frontMapNum, (*(iter - 1)).GetMapSize());
			(*iter).InitOutputLastStepDeltaKernel(frontMapNum, (*(iter - 1)).GetMapSize());//for adding momentum
			(*iter).InitBias(frontMapNum, CUDA_Algo_Lib::g_idx_iter_init_bias);
			(*iter).InitErros(batch_size);
			(*iter).InitOutputMaps(batch_size);
			break;
		default:
			break;
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::SetupTest(size_t batch_size)
{
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.begin();

	(*iter).InitOutputMaps(batch_size);
	iter++;
	for (iter; iter < vec_layers_.end(); iter++)
	{
		CUDA_Algo_Lib::g_idx_iter_init_bias = CUDA_Algo_Lib::g_idx_iter_init_bias + 1;

		size_t frontMapNum = (*(iter - 1)).GetOutMapNum();

		switch ((*iter).GetType())
		{
		case 'I':
			break;
		case 'C':
			// set map RectSize
			(*iter).SetMapSize((*(iter - 1)).GetMapSize().Substract((*iter).GetKernelSize(), 1));
			// initial convolution kernel_, quantities: frontMapNum*outMapNum_
			(*iter).InitKernel(frontMapNum);

			break;

		default:
			break;
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::BackPropagation(float* p_batch_data, float* p_batch_label)
{
	SetOutLayerErrors(p_batch_data, p_batch_label);
	SetHiddenLayerErrors();
}

void CUDA_Algo_Lib::CUDACNN::Forward(float* p_batch_data)
{
	SetInLayerOutput(p_batch_data);
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.begin()+1;
	//iter++;
	for (iter; iter < vec_layers_.end(); iter++)
	{
		switch ((*iter).GetType())
		{
		case 'C':
			SetConvOutput((*iter), (*(iter - 1)));
			break;
		case 'S':
			SetSampOutput((*iter), (*(iter - 1)));
			break;
		case 'H':
			SetFCHLayerOutput((*iter), (*(iter - 1)));
			break;
		case 'O':
			SetOutLayerOutput((*iter), (*(iter - 1)));
			break;
		default:
			break;
		}

	}
}

hipError_t CUDA_Algo_Lib::CUDACNN::SetInLayerOutput(float* p_batch_data)
{
	std::cout << "Execute CUDA_Algo_Lib::CUDACNN::SetInLayerOutput()" << std::endl;

	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.begin();

	RectSize map_size = (*iter).GetMapSize();
	size_t out_map_num = (*iter).GetOutMapNum();
	size_t dev_output_maps_size = batch_size_ * out_map_num * map_size.rows_ * map_size.cols_;

	////std::copy(p_batch_data, (p_batch_data + (batch_size_ * out_map_num * map_size.rows_ * map_size.cols_)), (*iter).vec_output_maps_.begin());
	memcpy((*iter).vec_output_maps_.data(), p_batch_data, dev_output_maps_size * sizeof(float));
	
	hipError_t cudaStatus;
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy((*iter).p_dev_output_maps_, (*iter).vec_output_maps_.data(),
		dev_output_maps_size * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}

	return cudaStatus;

}
// for change the value in m_Layers
hipError_t CUDA_Algo_Lib::CUDACNN::SetConvOutput(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer)
{
	std::cout << "Execute CUDA_Algo_Lib::CUDACNN::SetConvOutput()" << std::endl;
	
	size_t layer_map_num = r_layer.GetOutMapNum();
	size_t lastlayer_map_num = r_lastlayer.GetOutMapNum();
	size_t lastlayer_map_x = r_lastlayer.GetMapSize().rows_;
	size_t lastlayer_map_y = r_lastlayer.GetMapSize().cols_;
	size_t layer_kernel_x = r_layer.GetKernelSize().rows_;
	size_t layer_kernel_y = r_layer.GetKernelSize().cols_;
	size_t layer_map_x = r_layer.GetMapSize().rows_;
	size_t layer_map_y = r_layer.GetMapSize().cols_;
	size_t layer_map_size = layer_map_x * layer_map_y;
	std::vector<float> vec_sum(layer_map_size, 0.0);
	std::vector<float> vec_sum_now(layer_map_size, 0.0);
	
	float* p_dev_sums = nullptr;
	float* p_dev_sums_now = nullptr;

	hipError_t cudaStatus;
	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&p_dev_sums,
		(layer_map_size * sizeof(float)));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(p_dev_sums);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(p_dev_sums, vec_sum.data(),
		layer_map_size * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(p_dev_sums);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&p_dev_sums_now,
		(layer_map_size * sizeof(float)));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(p_dev_sums_now);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(p_dev_sums_now, vec_sum_now.data(),
		layer_map_size * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(p_dev_sums_now);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_lastlayer.p_dev_output_maps_, r_lastlayer.vec_output_maps_.data(),
		r_lastlayer.vec_output_maps_.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_layer.p_dev_kernel_, r_layer.vec_kernel_.data(),
		r_layer.vec_kernel_.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t i = 0; i < layer_map_num; i++)
		{
			for (size_t j = 0; j < lastlayer_map_num; j++)
			{
				size_t shift_idx_lastlayer_batch_map = idx_batch * lastlayer_map_num * lastlayer_map_x * lastlayer_map_y;
				size_t shift_idx_lastlayer_out_map = j * lastlayer_map_x * lastlayer_map_y;
				unsigned int dev_shift_idx_lastlayer_map = (unsigned int)(shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map);
				//float* p_lastlayer_map = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;

				size_t shift_idx_layer_front_kernel = j * layer_map_num * layer_kernel_x * layer_kernel_y;
				size_t shift_idx_layer_out_kernel = i * layer_kernel_x * layer_kernel_y;
				unsigned int dev_shift_idx_layer_kernel = (unsigned int)(shift_idx_layer_front_kernel + shift_idx_layer_out_kernel);
				//float* p_layer_kernel = r_layer.vec_kernel_.data() + shift_idx_layer_front_kernel + shift_idx_layer_out_kernel;

				size_t dev_num_conv_row = lastlayer_map_x - layer_kernel_x + 1;
				size_t dev_num_conv_col = lastlayer_map_y - layer_kernel_y + 1;

				dim3 dev_dim_grid_conv(dev_num_conv_row, dev_num_conv_col);
				dim3 dev_dim_block_conv(layer_kernel_x, layer_kernel_y);
				dim3 dev_dim_block_arrplus(layer_map_x, layer_map_y);

				if (j == 0)
				{
					
					//ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum.data());
					////each time we calculate one image of batch and also calculate relu 
					CUDAConvNValid <<<dev_dim_grid_conv, dev_dim_block_conv >>> (r_lastlayer.p_dev_output_maps_, r_layer.p_dev_kernel_, 
						dev_shift_idx_lastlayer_map, dev_shift_idx_layer_kernel, 
						(unsigned int)lastlayer_map_x, (unsigned int)lastlayer_map_y,
						p_dev_sums);

					// hipDeviceSynchronize waits for the kernel to finish, and returns
					// any errors encountered during the launch.
					cudaStatus = hipDeviceSynchronize();
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
						return cudaStatus;
					}

					// Copy output vector from GPU buffer to host memory.
					cudaStatus = hipMemcpy(vec_sum.data(), p_dev_sums, layer_map_size * sizeof(float), hipMemcpyDeviceToHost);
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipMemcpy failed!");
						return cudaStatus;
					}

				}
				else {
					//ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum_now.data());
					//CalConvArrayPlus(vec_sum_now.data(), vec_sum.data(), layer_map_x, layer_map_y);// sumNow 
					CUDAConvNValid << <dev_dim_grid_conv, dev_dim_block_conv >> > (r_lastlayer.p_dev_output_maps_, r_layer.p_dev_kernel_, 
						dev_shift_idx_lastlayer_map, dev_shift_idx_layer_kernel, 
						(unsigned int)lastlayer_map_x, (unsigned int)lastlayer_map_y,
						p_dev_sums_now);

					// hipDeviceSynchronize waits for the kernel to finish, and returns
					// any errors encountered during the launch.
					cudaStatus = hipDeviceSynchronize();
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
						return cudaStatus;
					}

					CUDACalConvArrayPlus << <1, dev_dim_block_arrplus >> > (p_dev_sums_now, p_dev_sums);

					// hipDeviceSynchronize waits for the kernel to finish, and returns
					// any errors encountered during the launch.
					cudaStatus = hipDeviceSynchronize();
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
						return cudaStatus;
					}

				}
			}

			//ActiveRelu(vec_sum.data(), r_layer.vec_bias_.at(i), layer_map_x, layer_map_y);//for relu active fun.

			dim3 dev_dim_block_active_relu(layer_map_x, layer_map_y);
			CUDAActiveRelu << <1, dev_dim_block_active_relu >> > (p_dev_sums, r_layer.vec_bias_.at(i));

			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
				return cudaStatus;
			}

			// Copy output vector from GPU buffer to host memory.
			cudaStatus = hipMemcpy(vec_sum.data(), p_dev_sums, 
				layer_map_size * sizeof(float), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				return cudaStatus;
			}

			size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
			size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
			float* p_layer_out_map = r_layer.vec_output_maps_.data() + shift_idx_layer_batch_map + shift_idx_layer_out_map;
			memcpy(p_layer_out_map, vec_sum.data(), (layer_map_x * layer_map_y * sizeof(float)));
		}
	}

	hipFree(p_dev_sums);
	hipFree(p_dev_sums_now);

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_layer.p_dev_output_maps_, r_layer.vec_output_maps_.data(),
		r_layer.vec_output_maps_.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}

	return cudaStatus;

}

hipError_t CUDA_Algo_Lib::CUDACNN::SetSampOutput(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer)
{
	std::cout << "Execute CUDA_Algo_Lib::CUDACNN::SetSampOutput()" << std::endl;

	size_t lastlayer_map_num = r_lastlayer.GetOutMapNum();
	size_t lastlayer_map_x = r_lastlayer.GetMapSize().rows_;
	size_t lastlayer_map_y = r_lastlayer.GetMapSize().cols_;
	size_t layer_map_x = r_layer.GetMapSize().rows_;
	size_t layer_map_y = r_layer.GetMapSize().cols_;
	size_t layer_map_size = layer_map_x * layer_map_y;
	RectSize scale_size = r_layer.GetScaleSize();
	std::vector<float> vec_samp_matrix(layer_map_size, 0.0);
	//float* p_lastlayer_map = NULL;

	float* p_dev_samp_matrix = nullptr;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&p_dev_samp_matrix,
		(layer_map_size * sizeof(float)));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(p_dev_samp_matrix);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(p_dev_samp_matrix, vec_samp_matrix.data(),
		layer_map_size * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(p_dev_samp_matrix);
		return cudaStatus;
	}

	size_t shift_idx_lastlayer_batch_map = 0;
	size_t shift_idx_lastlayer_out_map = 0;
	size_t dev_shift_idx_lastlayer_map = 0;
	size_t dev_lastlayer_map_x = lastlayer_map_x;
	size_t dev_lastlayer_map_y = lastlayer_map_y;

	float dev_total_sacle = (float)(scale_size.rows_ * scale_size.cols_);
	size_t dev_out_matrix_rows = lastlayer_map_x / scale_size.rows_;
	size_t dev_out_matrix_cols = lastlayer_map_y / scale_size.cols_;
	if (dev_out_matrix_rows * scale_size.rows_ != lastlayer_map_x || dev_out_matrix_cols * scale_size.cols_ != lastlayer_map_y)
	{
		std::cout << "scale can not divide by p_matrix";
		exit(0);
	}

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t i = 0; i < lastlayer_map_num; i++)
		{
			shift_idx_lastlayer_batch_map = idx_batch * lastlayer_map_num * lastlayer_map_x * lastlayer_map_y;
			shift_idx_lastlayer_out_map = i * lastlayer_map_x * lastlayer_map_y;
			dev_shift_idx_lastlayer_map = shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;
			//p_lastlayer_map = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;			
			//ScaleMatrix(p_lastlayer_map, scale_size, lastlayer_map_x, lastlayer_map_y, vec_samp_matrix.data());
			
			dim3 dev_dim_grid_scale(dev_out_matrix_rows, dev_out_matrix_cols);
			dim3 dev_dim_block_scale(scale_size.rows_, scale_size.cols_);
			CUDAScaleMatrix << <dev_dim_grid_scale, dev_dim_block_scale >> > (r_lastlayer.p_dev_output_maps_, (unsigned int)dev_shift_idx_lastlayer_map, (unsigned int)lastlayer_map_y, dev_total_sacle, p_dev_samp_matrix);

			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
				return cudaStatus;
			}

			// Copy output vector from GPU buffer to host memory.
			cudaStatus = hipMemcpy(vec_samp_matrix.data(), p_dev_samp_matrix,
				layer_map_size * sizeof(float), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				return cudaStatus;
			}

			size_t shift_idx_layer_batch_map = idx_batch * lastlayer_map_num * layer_map_x * layer_map_y;
			size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
			float* p_layer_out_map = r_layer.vec_output_maps_.data() + shift_idx_layer_batch_map + shift_idx_layer_out_map;
			memcpy(p_layer_out_map, vec_samp_matrix.data(), (layer_map_x * layer_map_y * sizeof(float)));
		}
	}

	hipFree(p_dev_samp_matrix);

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_layer.p_dev_output_maps_, r_layer.vec_output_maps_.data(),
		r_layer.vec_output_maps_.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}

	return cudaStatus;

}

hipError_t CUDA_Algo_Lib::CUDACNN::SetFCHLayerOutput(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer)
{
	std::cout << "Execute CUDA_Algo_Lib::CUDACNN::SetFCHLayerOutput()" << std::endl;

	size_t layer_map_num = r_layer.GetOutMapNum();
	size_t lastlayer_map_num = r_lastlayer.GetOutMapNum();
	size_t lastlayer_map_x = r_lastlayer.GetMapSize().rows_;
	size_t lastlayer_map_y = r_lastlayer.GetMapSize().cols_;
	size_t layer_kernel_x = r_layer.GetKernelSize().rows_;
	size_t layer_kernel_y = r_layer.GetKernelSize().cols_;
	size_t layer_map_x = r_layer.GetMapSize().rows_;
	size_t layer_map_y = r_layer.GetMapSize().cols_;
	size_t layer_map_size = layer_map_x * layer_map_y;
	std::vector<float> vec_sum(layer_map_x * layer_map_y, 0.0);
	std::vector<float> vec_sum_now(layer_map_x * layer_map_y, 0.0);

	float* p_dev_sums = nullptr;
	float* p_dev_sums_now = nullptr;

	hipError_t cudaStatus;
	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&p_dev_sums,
		(layer_map_size * sizeof(float)));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(p_dev_sums);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(p_dev_sums, vec_sum.data(),
		layer_map_size * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(p_dev_sums);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&p_dev_sums_now,
		(layer_map_size * sizeof(float)));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(p_dev_sums_now);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(p_dev_sums_now, vec_sum_now.data(),
		layer_map_size * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(p_dev_sums_now);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_lastlayer.p_dev_output_maps_, r_lastlayer.vec_output_maps_.data(),
		r_lastlayer.vec_output_maps_.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_layer.p_dev_kernel_, r_layer.vec_kernel_.data(),
		r_layer.vec_kernel_.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t i = 0; i < layer_map_num; i++)
		{
			for (size_t j = 0; j < lastlayer_map_num; j++)
			{
				size_t shift_idx_lastlayer_batch_map = idx_batch * lastlayer_map_num * lastlayer_map_x * lastlayer_map_y;
				size_t shift_idx_lastlayer_out_map = j * lastlayer_map_x * lastlayer_map_y;
				unsigned int dev_shift_idx_lastlayer_map = (unsigned int)(shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map);
				//float* p_lastlayer_map = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;
			
				size_t shift_idx_layer_front_kernel = j * layer_map_num * layer_kernel_x * layer_kernel_y;
				size_t shift_idx_layer_out_kernel = i * layer_kernel_x * layer_kernel_y;
				unsigned int dev_shift_idx_layer_kernel = (unsigned int)(shift_idx_layer_front_kernel + shift_idx_layer_out_kernel);
				//float* p_layer_kernel = r_layer.vec_kernel_.data() + shift_idx_layer_front_kernel + shift_idx_layer_out_kernel;

				size_t dev_num_conv_row = lastlayer_map_x - layer_kernel_x + 1;
				size_t dev_num_conv_col = lastlayer_map_y - layer_kernel_y + 1;

				dim3 dev_dim_grid_conv(dev_num_conv_row, dev_num_conv_col);
				dim3 dev_dim_block_conv(layer_kernel_x, layer_kernel_y);
				dim3 dev_dim_block_arrplus(layer_map_x, layer_map_y);

				if (j == 0)
				{
					//ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum.data());
					////each time we calculate one image of batch and also calculate relu 
					CUDAConvNValid << <dev_dim_grid_conv, dev_dim_block_conv >> > (r_lastlayer.p_dev_output_maps_, r_layer.p_dev_kernel_,
						dev_shift_idx_lastlayer_map, dev_shift_idx_layer_kernel,
						(unsigned int)lastlayer_map_x, (unsigned int)lastlayer_map_y,
						p_dev_sums);

					// hipDeviceSynchronize waits for the kernel to finish, and returns
					// any errors encountered during the launch.
					cudaStatus = hipDeviceSynchronize();
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
						return cudaStatus;
					}

					// Copy output vector from GPU buffer to host memory.
					cudaStatus = hipMemcpy(vec_sum.data(), p_dev_sums, layer_map_size * sizeof(float), hipMemcpyDeviceToHost);
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipMemcpy failed!");
						return cudaStatus;
					}
				}
				else {
					//ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum_now.data());
					//CalFCHArrayPlus(vec_sum_now.data(), vec_sum.data(), layer_map_x, layer_map_y);// sumNow 

					CUDAConvNValid << <dev_dim_grid_conv, dev_dim_block_conv >> > (r_lastlayer.p_dev_output_maps_, r_layer.p_dev_kernel_,
						dev_shift_idx_lastlayer_map, dev_shift_idx_layer_kernel,
						(unsigned int)lastlayer_map_x, (unsigned int)lastlayer_map_y,
						p_dev_sums_now);

					// hipDeviceSynchronize waits for the kernel to finish, and returns
					// any errors encountered during the launch.
					cudaStatus = hipDeviceSynchronize();
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
						return cudaStatus;
					}

					CUDACalConvArrayPlus << <1, dev_dim_block_arrplus >> > (p_dev_sums_now, p_dev_sums);

					// hipDeviceSynchronize waits for the kernel to finish, and returns
					// any errors encountered during the launch.
					cudaStatus = hipDeviceSynchronize();
					if (cudaStatus != hipSuccess) {
						fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
						return cudaStatus;
					}

				}
			}

			////printf("ActiveRelu");
			//ActiveRelu(vec_sum.data(), r_layer.vec_bias_.at(i), layer_map_x, layer_map_y);//for relu active fun.

			dim3 dev_dim_block_active_relu(layer_map_x, layer_map_y);
			CUDAActiveRelu << <1, dev_dim_block_active_relu >> > (p_dev_sums, r_layer.vec_bias_.at(i));

			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
				return cudaStatus;
			}

			// Copy output vector from GPU buffer to host memory.
			cudaStatus = hipMemcpy(vec_sum.data(), p_dev_sums,
				layer_map_size * sizeof(float), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
				return cudaStatus;
			}

			//SetValue(r_layer.outputmaps_[idx_batch][i], sum, layer_map_x, layer_map_y);
			size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
			size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
			float* p_layer_out_map = r_layer.vec_output_maps_.data() + shift_idx_layer_batch_map + shift_idx_layer_out_map;
			memcpy(p_layer_out_map, vec_sum.data(), (layer_map_x * layer_map_y * sizeof(float)));

		}

	}

	hipFree(p_dev_sums);
	hipFree(p_dev_sums_now);

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_layer.p_dev_output_maps_, r_layer.vec_output_maps_.data(),
		r_layer.vec_output_maps_.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}

	return cudaStatus;

}

hipError_t CUDA_Algo_Lib::CUDACNN::SetOutLayerOutput(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer)
{
	std::cout << "Execute CUDA_Algo_Lib::CUDACNN::SetOutLayerOutput()" << std::endl;

	size_t layer_map_num = r_layer.GetOutMapNum();
	size_t lastlayer_map_num = r_lastlayer.GetOutMapNum();
	size_t lastlayer_map_x = r_lastlayer.GetMapSize().rows_;
	size_t lastlayer_map_y = r_lastlayer.GetMapSize().cols_;
	size_t layer_kernel_x = r_layer.GetKernelSize().rows_;
	size_t layer_kernel_y = r_layer.GetKernelSize().cols_;
	size_t layer_map_x = r_layer.GetMapSize().rows_;
	size_t layer_map_y = r_layer.GetMapSize().cols_;
	size_t layer_map_size = layer_map_x * layer_map_y;
	std::vector<float> vec_sum(layer_map_x * layer_map_y, 0.0);
	std::vector<float> vec_sum_now(layer_map_x * layer_map_y, 0.0);
	std::vector<float> vec_sum_expone(batch_size_, 0.0);

	float* p_dev_sums = nullptr;
	float* p_dev_sums_now = nullptr;
	float* p_dev_sums_expone = nullptr;

	hipError_t cudaStatus;
	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&p_dev_sums,
		(layer_map_size * sizeof(float)));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto CUDA_ERROR;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(p_dev_sums, vec_sum.data(),
		layer_map_size * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CUDA_ERROR;
	}

	cudaStatus = hipMalloc((void**)&p_dev_sums_now,
		(layer_map_size * sizeof(float)));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto CUDA_ERROR;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(p_dev_sums_now, vec_sum_now.data(),
		layer_map_size * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CUDA_ERROR;
	}

	cudaStatus = hipMalloc((void**)&p_dev_sums_expone,
		(batch_size_ * sizeof(float)));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto CUDA_ERROR;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(p_dev_sums_expone, vec_sum_expone.data(),
		vec_sum_expone.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CUDA_ERROR;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_lastlayer.p_dev_output_maps_, r_lastlayer.vec_output_maps_.data(),
		r_lastlayer.vec_output_maps_.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CUDA_ERROR;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_layer.p_dev_kernel_, r_layer.vec_kernel_.data(),
		r_layer.vec_kernel_.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CUDA_ERROR;
	}

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		//printf("ActiveRelu+softmax");
		//std::cout << "NO.of Batch: " << idx_batch << std::endl;
		for (size_t i = 0; i < layer_map_num; i++)
		{
			for (size_t j = 0; j < lastlayer_map_num; j++)
			{
				size_t shift_idx_lastlayer_batch_map = idx_batch * lastlayer_map_num * lastlayer_map_x * lastlayer_map_y;
				size_t shift_idx_lastlayer_out_map = j * lastlayer_map_x * lastlayer_map_y;
				float* p_lastlayer_map = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;

				size_t shift_idx_layer_front_kernel = j * layer_map_num * layer_kernel_x * layer_kernel_y;
				size_t shift_idx_layer_out_kernel = i * layer_kernel_x * layer_kernel_y;
				float* p_layer_kernel = r_layer.vec_kernel_.data() + shift_idx_layer_front_kernel + shift_idx_layer_out_kernel;

				if (j == 0)
				{
					ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum.data());
					//each time we calculate one image of batch and also calculate relu 

				}
				else {
					ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum_now.data());
					CalFCHArrayPlus(vec_sum_now.data(), vec_sum.data(), layer_map_x, layer_map_y);// sumNow 

				}
			}

			CalExpone(vec_sum.data(), r_layer.vec_bias_.at(i), layer_map_x, layer_map_y);

			size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
			size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
			float* p_layer_out_map = r_layer.vec_output_maps_.data() + shift_idx_layer_batch_map + shift_idx_layer_out_map;
			memcpy(p_layer_out_map, vec_sum.data(), (layer_map_x * layer_map_y * sizeof(float)));

		}

		for (size_t i = 0; i < layer_map_num; i++)
		{
			for (size_t ii = 0; ii < layer_map_x; ii++)
			{
				for (size_t jj = 0; jj < layer_map_y; jj++)
				{
					size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
					size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
					size_t shift_idx_layer_out_map_row = ii * layer_map_y;
					size_t idx_layer_out_map = shift_idx_layer_batch_map + shift_idx_layer_out_map + shift_idx_layer_out_map_row + jj;
					vec_sum_expone[idx_batch] += r_layer.vec_output_maps_.at(idx_layer_out_map);
				}
			}
		}

		for (size_t i = 0; i < layer_map_num; i++)
		{
			for (size_t ii = 0; ii < layer_map_x; ii++)
			{
				for (size_t jj = 0; jj < layer_map_y; jj++)
				{

					size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
					size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
					size_t shift_idx_layer_out_map_row = ii * layer_map_y;
					size_t idx_layer_out_map = shift_idx_layer_batch_map + shift_idx_layer_out_map + shift_idx_layer_out_map_row + jj;
					r_layer.vec_output_maps_[idx_layer_out_map] = r_layer.vec_output_maps_[idx_layer_out_map] / vec_sum_expone[idx_batch];

					std::cout << "Outputlayer's Softmax actual output(r_layer.outputmaps_[" << idx_batch << "][" << i << "][" << ii << "][" << jj << "]): " << r_layer.vec_output_maps_[idx_layer_out_map] << std::endl;
				}
			}
		}
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(r_layer.p_dev_output_maps_, r_layer.vec_output_maps_.data(),
		r_layer.vec_output_maps_.size() * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CUDA_ERROR;
	}

CUDA_ERROR:
	hipFree(p_dev_sums);
	hipFree(p_dev_sums_now);
	hipFree(p_dev_sums_expone);

	return cudaStatus;

}

//// ReLU+Softmax function
//hipError_t CUDA_Algo_Lib::CUDACNN::SetOutLayerOutput(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer)
//{
//	std::cout << "Execute CUDA_Algo_Lib::CUDACNN::SetOutLayerOutput()" << std::endl;
//
//	size_t layer_map_num = r_layer.GetOutMapNum();
//	size_t lastlayer_map_num = r_lastlayer.GetOutMapNum();
//	size_t lastlayer_map_x = r_lastlayer.GetMapSize().rows_;
//	size_t lastlayer_map_y = r_lastlayer.GetMapSize().cols_;
//	size_t layer_kernel_x = r_layer.GetKernelSize().rows_;
//	size_t layer_kernel_y = r_layer.GetKernelSize().cols_;
//	size_t layer_map_x = r_layer.GetMapSize().rows_;
//	size_t layer_map_y = r_layer.GetMapSize().cols_;
//	size_t layer_map_size = layer_map_x * layer_map_y;
//	std::vector<float> vec_sum(layer_map_x * layer_map_y, 0.0);
//	std::vector<float> vec_sum_now(layer_map_x * layer_map_y, 0.0);
//	std::vector<float> vec_sum_expone(batch_size_, 0.0);
//
//	float* p_dev_sums = nullptr;
//	float* p_dev_sums_now = nullptr;
//	float* p_dev_sums_expone = nullptr;
//
//	hipError_t cudaStatus;
//	// Allocate GPU buffers
//	cudaStatus = hipMalloc((void**)&p_dev_sums,
//		(layer_map_size * sizeof(float)));
//
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto CUDA_ERROR;
//	}
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = hipMemcpy(p_dev_sums, vec_sum.data(),
//		layer_map_size * sizeof(float), hipMemcpyHostToDevice);
//
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto CUDA_ERROR;
//	}
//
//	cudaStatus = hipMalloc((void**)&p_dev_sums_now,
//		(layer_map_size * sizeof(float)));
//
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto CUDA_ERROR;
//	}
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = hipMemcpy(p_dev_sums_now, vec_sum_now.data(),
//		layer_map_size * sizeof(float), hipMemcpyHostToDevice);
//
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto CUDA_ERROR;
//	}
//
//	cudaStatus = hipMalloc((void**)&p_dev_sums_expone,
//		(batch_size_ * sizeof(float)));
//
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto CUDA_ERROR;
//	}
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = hipMemcpy(p_dev_sums_expone, vec_sum_expone.data(),
//		vec_sum_expone.size() * sizeof(float), hipMemcpyHostToDevice);
//
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto CUDA_ERROR;
//	}
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = hipMemcpy(r_lastlayer.p_dev_output_maps_, r_lastlayer.vec_output_maps_.data(),
//		r_lastlayer.vec_output_maps_.size() * sizeof(float), hipMemcpyHostToDevice);
//
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto CUDA_ERROR;
//	}
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = hipMemcpy(r_layer.p_dev_kernel_, r_layer.vec_kernel_.data(),
//		r_layer.vec_kernel_.size() * sizeof(float), hipMemcpyHostToDevice);
//
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto CUDA_ERROR;
//	}
//
//
//	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
//	{
//		//printf("ActiveRelu+softmax");
//		//std::cout << "NO.of Batch: " << idx_batch << std::endl;
//		for (size_t i = 0; i < layer_map_num; i++)
//		{
//			for (size_t j = 0; j < lastlayer_map_num; j++)
//			{
//				size_t shift_idx_lastlayer_batch_map = idx_batch * lastlayer_map_num * lastlayer_map_x * lastlayer_map_y;
//				size_t shift_idx_lastlayer_out_map = j * lastlayer_map_x * lastlayer_map_y;
//				unsigned int dev_shift_idx_lastlayer_map = (unsigned int)(shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map);
//				float* p_lastlayer_map = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;
//			
//				size_t shift_idx_layer_front_kernel = j * layer_map_num * layer_kernel_x * layer_kernel_y;
//				size_t shift_idx_layer_out_kernel = i * layer_kernel_x * layer_kernel_y;
//				unsigned int dev_shift_idx_layer_kernel = (unsigned int)(shift_idx_layer_front_kernel + shift_idx_layer_out_kernel);
//				float* p_layer_kernel = r_layer.vec_kernel_.data() + shift_idx_layer_front_kernel + shift_idx_layer_out_kernel;
//
//				size_t dev_num_conv_row = lastlayer_map_x - layer_kernel_x + 1;
//				size_t dev_num_conv_col = lastlayer_map_y - layer_kernel_y + 1;
//
//				dim3 dev_dim_grid_conv(dev_num_conv_row, dev_num_conv_col);
//				dim3 dev_dim_block_conv(layer_kernel_x, layer_kernel_y);
//				dim3 dev_dim_block_arrplus(layer_map_x, layer_map_y);
//
//				if (j == 0)
//				{
//					//ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum.data());
//					
//					//each time we calculate one image of batch and also calculate relu 
//					CUDAConvNValid << <dev_dim_grid_conv, dev_dim_block_conv >> > (r_lastlayer.p_dev_output_maps_, r_layer.p_dev_kernel_,
//						dev_shift_idx_lastlayer_map, dev_shift_idx_layer_kernel,
//						(unsigned int)lastlayer_map_x, (unsigned int)lastlayer_map_y,
//						p_dev_sums);
//
//					// hipDeviceSynchronize waits for the kernel to finish, and returns
//					// any errors encountered during the launch.
//					cudaStatus = hipDeviceSynchronize();
//					if (cudaStatus != hipSuccess) {
//						fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//						goto CUDA_ERROR;
//					}
//				}
//				else {
//					//ConvNValid(p_lastlayer_map, p_layer_kernel, lastlayer_map_x, lastlayer_map_y, layer_kernel_x, layer_kernel_y, vec_sum_now.data());
//					//CalFCHArrayPlus(vec_sum_now.data(), vec_sum.data(), layer_map_x, layer_map_y);// sumNow 
//
//					CUDAConvNValid << <dev_dim_grid_conv, dev_dim_block_conv >> > (r_lastlayer.p_dev_output_maps_, r_layer.p_dev_kernel_,
//						dev_shift_idx_lastlayer_map, dev_shift_idx_layer_kernel,
//						(unsigned int)lastlayer_map_x, (unsigned int)lastlayer_map_y,
//						p_dev_sums_now);
//
//					// hipDeviceSynchronize waits for the kernel to finish, and returns
//					// any errors encountered during the launch.
//					cudaStatus = hipDeviceSynchronize();
//					if (cudaStatus != hipSuccess) {
//						fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//						goto CUDA_ERROR;
//					}
//
//					CUDACalConvArrayPlus << <1, dev_dim_block_arrplus >> > (p_dev_sums_now, p_dev_sums);
//
//					// hipDeviceSynchronize waits for the kernel to finish, and returns
//					// any errors encountered during the launch.
//					cudaStatus = hipDeviceSynchronize();
//					if (cudaStatus != hipSuccess) {
//						fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//						goto CUDA_ERROR;
//					}
//				}
//			}
//
//			//cudaStatus = hipMemcpy(vec_sum.data(), p_dev_sums,
//			//	vec_sum.size() * sizeof(float), hipMemcpyDeviceToHost);
//			//if (cudaStatus != hipSuccess) {
//			//	fprintf(stderr, "hipMemcpy failed!");
//			//	return cudaStatus;
//			//}
//
//			//CalExpone(vec_sum.data(), r_layer.vec_bias_.at(i), layer_map_x, layer_map_y);
//
//			//cudaStatus = hipMemcpy(p_dev_sums, vec_sum.data(),
//			//	vec_sum.size() * sizeof(float), hipMemcpyHostToDevice);
//			//if (cudaStatus != hipSuccess) {
//			//	fprintf(stderr, "hipMemcpy failed!");
//			//	return cudaStatus;
//			//}
//
//			dim3 dev_dim_block_expone(layer_map_x, layer_map_y);
//			CUDACalExpone << <1, dev_dim_block_expone >> > (p_dev_sums, r_layer.vec_bias_.at(i));
//			
//			// hipDeviceSynchronize waits for the kernel to finish, and returns
//			// any errors encountered during the launch.
//			cudaStatus = hipDeviceSynchronize();
//			if (cudaStatus != hipSuccess) {
//				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//				goto CUDA_ERROR;
//			}
//
//			// Copy output vector from GPU buffer to host memory.
//			cudaStatus = hipMemcpy(vec_sum.data(), p_dev_sums,
//				vec_sum.size() * sizeof(float), hipMemcpyDeviceToHost);
//			if (cudaStatus != hipSuccess) {
//				fprintf(stderr, "hipMemcpy failed!");
//				return cudaStatus;
//			}
//
//			size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
//			size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
//			unsigned int dev_shift_idx_layer_map = (unsigned int)(shift_idx_layer_batch_map + shift_idx_layer_out_map);
//			float* p_layer_out_map = r_layer.vec_output_maps_.data() + shift_idx_layer_batch_map + shift_idx_layer_out_map;
//			memcpy(p_layer_out_map, vec_sum.data(), (layer_map_x * layer_map_y * sizeof(float)));
//
//			//// Copy output vector from GPU buffer to GPU buffer.
//			//dim3 dev_dim_block_shift_assign(layer_map_x, layer_map_y);
//			//CUDAShiftAssignValue << <1, dev_dim_block_shift_assign >> > (r_layer.p_dev_output_maps_, dev_shift_idx_layer_map, p_dev_sums);
//			//
//			//// hipDeviceSynchronize waits for the kernel to finish, and returns
//			//// any errors encountered during the launch.
//			//cudaStatus = hipDeviceSynchronize();
//			//if (cudaStatus != hipSuccess) {
//			//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//			//	goto CUDA_ERROR;
//			//}
//
//		}
//
//		for (size_t i = 0; i < layer_map_num; i++)
//		{
//			for (size_t ii = 0; ii < layer_map_x; ii++)
//			{
//				for (size_t jj = 0; jj < layer_map_y; jj++)
//				{
//					size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
//					size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
//					size_t shift_idx_layer_out_map_row = ii * layer_map_y;
//					size_t idx_layer_out_map = shift_idx_layer_batch_map + shift_idx_layer_out_map + shift_idx_layer_out_map_row + jj;
//					vec_sum_expone[idx_batch] += r_layer.vec_output_maps_.at(idx_layer_out_map);
//				}
//			}
//		}
//
//		for (size_t i = 0; i < layer_map_num; i++)
//		{
//			for (size_t ii = 0; ii < layer_map_x; ii++)
//			{
//				for (size_t jj = 0; jj < layer_map_y; jj++)
//				{
//
//					size_t shift_idx_layer_batch_map = idx_batch * layer_map_num * layer_map_x * layer_map_y;
//					size_t shift_idx_layer_out_map = i * layer_map_x * layer_map_y;
//					size_t shift_idx_layer_out_map_row = ii * layer_map_y;
//					size_t idx_layer_out_map = shift_idx_layer_batch_map + shift_idx_layer_out_map + shift_idx_layer_out_map_row + jj;
//					r_layer.vec_output_maps_[idx_layer_out_map] = r_layer.vec_output_maps_[idx_layer_out_map] / vec_sum_expone[idx_batch];
//
//					std::cout << "Outputlayer's Softmax actual output(r_layer.outputmaps_[" << idx_batch << "][" << i << "][" << ii << "][" << jj << "]): " << r_layer.vec_output_maps_[idx_layer_out_map] << std::endl;
//				}
//			}
//		}
//
//		//dim3 dev_dim_grid_sum_expone(1, layer_map_num);
//		//dim3 dev_dim_block_sum_expone(layer_map_x, layer_map_y);
//		//
//		//CUDACalSumExpone <<<dev_dim_grid_sum_expone, dev_dim_block_sum_expone >>> (p_dev_sums_expone, r_layer.p_dev_output_maps_, (unsigned int)idx_batch);
//		//cudaStatus = hipDeviceSynchronize();
//		//if (cudaStatus != hipSuccess) {
//		//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//		//	goto CUDA_ERROR;
//		//}
//
//		//// Copy output vector from GPU buffer to host memory.
//		//cudaStatus = hipMemcpy(vec_sum_expone.data(), p_dev_sums_expone,
//		//	vec_sum_expone.size() * sizeof(float), hipMemcpyDeviceToHost);
//		//std::cout << vec_sum_expone.at(0) << ", " << vec_sum_expone.at(1) << std::endl;
//		//if (cudaStatus != hipSuccess) {
//		//	fprintf(stderr, "hipMemcpy failed!");
//		//	return cudaStatus;
//		//}
//
//		//dim3 dev_dim_grid_softmax(1, layer_map_num);
//		//dim3 dev_dim_block_softmax(layer_map_x, layer_map_y);
//
//		//CUDACalSoftmax <<<dev_dim_grid_softmax, dev_dim_block_softmax >>> (r_layer.p_dev_output_maps_, p_dev_sums_expone, idx_batch);
//		//cudaStatus = hipDeviceSynchronize();
//		//if (cudaStatus != hipSuccess) {
//		//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//		//	goto CUDA_ERROR;
//		//}
//
//		//// Copy output vector from GPU buffer to host memory.
//		//cudaStatus = hipMemcpy(r_layer.vec_output_maps_.data(), r_layer.p_dev_output_maps_,
//		//	r_layer.vec_output_maps_.size() * sizeof(float), hipMemcpyDeviceToHost);
//		//if (cudaStatus != hipSuccess) {
//		//	fprintf(stderr, "hipMemcpy failed!");
//		//	return cudaStatus;
//		//}
//
//	}
//
//	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = hipMemcpy(r_layer.p_dev_output_maps_, r_layer.vec_output_maps_.data(),
//		r_layer.vec_output_maps_.size() * sizeof(float), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto CUDA_ERROR;
//	}
//
//CUDA_ERROR:
//	hipFree(p_dev_sums);
//	hipFree(p_dev_sums_now);
//	hipFree(p_dev_sums_expone);
//
//	return cudaStatus;
//}

void CUDA_Algo_Lib::CUDACNN::SetOutLayerErrors(float* p_input_maps, float* p_target_labels)
{
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.end();
	iter--;
	size_t layer_outmap_num = (*iter).GetOutMapNum();
	float mean_error = 0.0, max_error = 0.0;

	//FILE* fy;
	//fy = fopen("./outputdata/error.txt", "a");

	////if( (err=fopen_s(&fy, "error.txt", "a")) != 0 )
	////	exit(1) ;

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t idx_map = 0; idx_map < layer_outmap_num; idx_map++)
		{
			//float val_out_map = (*iter).outputmaps_[idx_batch][idx_map][0][0];
			float val_target_label = p_target_labels[idx_batch * layer_outmap_num + idx_map];
			size_t shift_idx_layer_batch_map = idx_batch * layer_outmap_num * ((*iter).GetMapSize().rows_) * ((*iter).GetMapSize().cols_);
			size_t shift_idx_layer_out_map = idx_map * ((*iter).GetMapSize().rows_) * ((*iter).GetMapSize().cols_);
			size_t shift_idx_layer_out_map_row = 0 * ((*iter).GetMapSize().cols_);
			size_t idx_layer_out_map = shift_idx_layer_batch_map + shift_idx_layer_out_map + shift_idx_layer_out_map_row + 0;
			float val_out_map = (*iter).vec_output_maps_.at(idx_layer_out_map);

			//printf("Cross-entropy cost function for ReLU+Softmax");
			//Cross entropy for softmax form
			(*iter).SetError(idx_batch, idx_map, 0, 0, (val_out_map - val_target_label));
			mean_error = abs(val_out_map - val_target_label);

			//fprintf(fy, "%f ", mean_error);
			//// 			mean_error += abs(val_target_label-val_out_map);
			//// 			if (abs(val_target_label-val_out_map)>max_error)
			//// 			{
			//// 				max_error = abs(val_target_label-val_out_map);
			//// 			}
		}
		//fprintf(fy, "\n");
	}
	//fprintf(fy, "\n");
	//fclose(fy);
	//// 	std::cout<<"Mean error of each mini batch: "<<mean_error<<std::endl;
	//// 	std::cout<<"The max error of one output in mini batch: "<<max_error<<std::endl;
}

hipError_t  CUDA_Algo_Lib::CUDACNN::SetFCHiddenLayerErrors(CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer, CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_nextlayer)//for add FC hiddenlayer
{
	size_t lastlayer_outmap_num = r_lastlayer.GetOutMapNum();
	size_t layer_batch_outmaps_size = r_layer.vec_output_maps_.size();
	size_t layer_outmap_num = r_layer.GetOutMapNum();
	size_t layer_outmap_rows = r_layer.GetMapSize().rows_;
	size_t layer_outmap_cols = r_layer.GetMapSize().cols_;
	size_t layer_each_outmap_size = layer_outmap_rows * layer_outmap_cols;
	size_t nextlayer_outmap_num = r_nextlayer.GetOutMapNum();
	size_t nextlayer_outmap_rows = r_nextlayer.GetMapSize().rows_;
	size_t nextlayer_outmap_cols = r_nextlayer.GetMapSize().cols_;
	size_t nextlayer_kernel_rows = r_nextlayer.GetKernelSize().rows_;
	size_t nextlayer_kernel_cols = r_nextlayer.GetKernelSize().cols_;

	std::vector<float> vec_layer_derivative_active_fun;
	vec_layer_derivative_active_fun.reserve(layer_batch_outmaps_size);
	vec_layer_derivative_active_fun.resize(layer_batch_outmaps_size);

	std::vector<float> vec_layer_sum_righttern_local_gradient;
	vec_layer_sum_righttern_local_gradient.reserve(layer_batch_outmaps_size);
	vec_layer_sum_righttern_local_gradient.resize(layer_batch_outmaps_size);

	float* p_dev_layer_deriv_active_funcs = nullptr;
	float* p_dev_layer_sum_righttern_local_grads = nullptr;

	hipError_t cudaStatus;
	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&p_dev_layer_deriv_active_funcs,
		(layer_batch_outmaps_size * sizeof(float)));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto CUDA_ERROR_SetFCHiddenLayerErrors;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(p_dev_layer_deriv_active_funcs, vec_layer_derivative_active_fun.data(),
		layer_batch_outmaps_size * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CUDA_ERROR_SetFCHiddenLayerErrors;
	}

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&p_dev_layer_sum_righttern_local_grads,
		(layer_batch_outmaps_size * sizeof(float)));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto CUDA_ERROR_SetFCHiddenLayerErrors;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(p_dev_layer_sum_righttern_local_grads, vec_layer_sum_righttern_local_gradient.data(),
		layer_batch_outmaps_size * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CUDA_ERROR_SetFCHiddenLayerErrors;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_layer.p_dev_output_maps_, r_layer.vec_output_maps_.data(),
		r_layer.vec_output_maps_.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CUDA_ERROR_SetFCHiddenLayerErrors;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_nextlayer.p_dev_kernel_, r_nextlayer.vec_kernel_.data(),
		r_nextlayer.vec_kernel_.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CUDA_ERROR_SetFCHiddenLayerErrors;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_nextlayer.p_dev_errors_, r_nextlayer.vec_errors_.data(),
		r_nextlayer.vec_errors_.size() * sizeof(float), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CUDA_ERROR_SetFCHiddenLayerErrors;
	}

	//printf("================================================================================\n");
	//printf("derivative active functions' value");
	dim3 dev_dim_grid_deriv_active_funcs(batch_size_, layer_outmap_num);
	dim3 dev_dim_block_deriv_active_funcs(layer_outmap_rows, layer_outmap_cols);

	CUDACalDerivActiveReLUFCH <<<dev_dim_grid_deriv_active_funcs, dev_dim_block_deriv_active_funcs >>> (
		p_dev_layer_deriv_active_funcs, r_layer.p_dev_output_maps_);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto CUDA_ERROR_SetFCHiddenLayerErrors;
	}

	//printf("================================================================================\n");

	dim3 dev_dim_grid_sum_righttern_local_grads(batch_size_, layer_outmap_num);
	dim3 dev_dim_block_sum_righttern_local_grads(nextlayer_outmap_num);

	CUDACalSumRightTernLocalGradientFCH <<< dev_dim_grid_sum_righttern_local_grads, dev_dim_block_sum_righttern_local_grads >>> (
		p_dev_layer_sum_righttern_local_grads, r_nextlayer.p_dev_errors_, r_nextlayer.p_dev_kernel_);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto CUDA_ERROR_SetFCHiddenLayerErrors;
	}

	dim3 dev_dim_grid_local_grads(batch_size_, layer_outmap_num);
	dim3 dev_dim_block_local_grads(layer_outmap_rows, layer_outmap_cols);

	CUDACalElementwiseMultiplication << < dev_dim_grid_local_grads, dev_dim_block_local_grads >> > (
		r_layer.p_dev_errors_, p_dev_layer_deriv_active_funcs, p_dev_layer_sum_righttern_local_grads);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto CUDA_ERROR_SetFCHiddenLayerErrors;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(r_layer.vec_errors_.data(), r_layer.p_dev_errors_,
		r_layer.vec_errors_.size() * sizeof(float), hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CUDA_ERROR_SetFCHiddenLayerErrors;
	}

CUDA_ERROR_SetFCHiddenLayerErrors:
	hipFree(p_dev_layer_deriv_active_funcs);
	hipFree(p_dev_layer_sum_righttern_local_grads);

	return cudaStatus;
}

void CUDA_Algo_Lib::CUDACNN::SetHiddenLayerErrors()
{
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.end();
	iter = iter - 2;
	for (iter; iter > vec_layers_.begin(); iter--)
	{
		switch ((*(iter)).GetType())
		{
		case 'C':
			SetConvErrors((*iter), (*(iter + 1)));
			break;
		case 'S':
			SetSampErrors((*iter), (*(iter + 1)));
			break;
		case 'H':
			SetFCHiddenLayerErrors((*(iter - 1)), (*iter), (*(iter + 1)));
			break;
		default:
			break;
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::SetSampErrors(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_nextlayer)
{
	size_t layer_outmap_num = r_layer.GetOutMapNum();
	size_t layer_outmap_rows = r_layer.GetMapSize().rows_;
	size_t layer_outmap_cols = r_layer.GetMapSize().cols_;
	size_t nextlayer_outmap_num = r_nextlayer.GetOutMapNum();
	size_t nextlayer_outmap_rows = r_nextlayer.GetMapSize().rows_;
	size_t nextlayer_outmap_cols = r_nextlayer.GetMapSize().cols_;
	size_t nextlayer_kernel_rows = r_nextlayer.GetKernelSize().rows_;
	size_t nextlayer_kernel_cols = r_nextlayer.GetKernelSize().cols_;

	float* p_nextlayer_error = nullptr;
	float* p_nextlayer_kernel = nullptr;
	std::vector<float> vec_sum(layer_outmap_rows * layer_outmap_cols, 0.0);
	std::vector<float> vec_sum_now(layer_outmap_rows * layer_outmap_cols, 0.0);
	std::vector<float> vec_rot_matrix(nextlayer_kernel_rows * nextlayer_kernel_cols, 0.0);
	std::vector<float> vec_nextlayer_extend_matrix((nextlayer_outmap_rows+2*(nextlayer_kernel_rows-1)) * (nextlayer_outmap_cols+2*(nextlayer_kernel_cols-1)), 0.0);

	//calculate
	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
		{
			for (size_t idx_nextlayer_outmap = 0; idx_nextlayer_outmap < nextlayer_outmap_num; idx_nextlayer_outmap++)
			{

				p_nextlayer_error = r_nextlayer.GetError(idx_batch, idx_nextlayer_outmap);
				p_nextlayer_kernel = r_nextlayer.GetKernel(idx_layer_outmap, idx_nextlayer_outmap);
				if (idx_nextlayer_outmap == 0)
				{
					Rot180(p_nextlayer_kernel, nextlayer_kernel_rows, nextlayer_kernel_cols, vec_rot_matrix.data());
					ConvNSampFull(p_nextlayer_error, vec_rot_matrix.data(), nextlayer_outmap_rows, nextlayer_outmap_cols, nextlayer_kernel_rows, nextlayer_kernel_cols, vec_sum.data(), vec_nextlayer_extend_matrix.data());

				}
				else
				{
					Rot180(p_nextlayer_kernel, nextlayer_kernel_rows, nextlayer_kernel_cols, vec_rot_matrix.data());
					ConvNSampFull(p_nextlayer_error, vec_rot_matrix.data(), nextlayer_outmap_rows, nextlayer_outmap_cols, nextlayer_kernel_rows, nextlayer_kernel_cols, vec_sum_now.data(), vec_nextlayer_extend_matrix.data());
					CalSampArrayPlus(vec_sum_now.data(), vec_sum.data(), layer_outmap_rows, layer_outmap_cols);

				}

			}
			r_layer.SetSampLayerError(idx_batch, idx_layer_outmap, vec_sum.data(), layer_outmap_rows, layer_outmap_cols);
		}
	}

}

void CUDA_Algo_Lib::CUDACNN::SetConvErrors(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_nextlayer)
{
	size_t layer_outmap_num = r_layer.GetOutMapNum();
	size_t layer_outmap_rows = r_layer.GetMapSize().rows_;
	size_t layer_outmap_cols = r_layer.GetMapSize().cols_;
	size_t nextlayer_outmap_rows = r_nextlayer.GetMapSize().rows_;
	size_t nextlayer_outmap_cols = r_nextlayer.GetMapSize().cols_;

	float* p_nextlayer_error = nullptr;
	float* p_layer_outmap = nullptr;
	std::vector<float> vec_layer_outmatrix;
	std::vector<float> vec_layer_outkroneckermatrix;
	vec_layer_outmatrix.reserve(layer_outmap_rows * layer_outmap_cols);
	vec_layer_outkroneckermatrix.reserve(layer_outmap_rows * layer_outmap_cols);
	vec_layer_outmatrix.resize(layer_outmap_rows * layer_outmap_cols);
	vec_layer_outkroneckermatrix.resize(layer_outmap_rows * layer_outmap_cols);
	RectSize layer_scale_size = r_layer.GetScaleSize();

	for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
	{
		for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
		{
			layer_scale_size = r_nextlayer.GetScaleSize();
			p_nextlayer_error = r_nextlayer.GetError(idx_batch, idx_layer_outmap);
			size_t shift_idx_layer_batch_map = idx_batch * layer_outmap_num * layer_outmap_rows * layer_outmap_cols;
			size_t shift_idx_layer_out_map = idx_layer_outmap * layer_outmap_rows * layer_outmap_cols;
			p_layer_outmap = r_layer.vec_output_maps_.data() + shift_idx_layer_batch_map + shift_idx_layer_out_map;

			//printf("derivative of ReLu");
			//derivative of ReLu
			MatrixDreluConv(p_layer_outmap, layer_outmap_rows, layer_outmap_cols, vec_layer_outmatrix.data());//for relu active fun.

			CalKronecker(p_nextlayer_error, layer_scale_size, nextlayer_outmap_rows, nextlayer_outmap_cols, vec_layer_outkroneckermatrix.data(), layer_outmap_rows, layer_outmap_cols);
			CalMatrixMultiply(vec_layer_outmatrix.data(), vec_layer_outkroneckermatrix.data(), layer_outmap_rows, layer_outmap_cols);

			r_layer.SetConvLayerError(idx_batch, idx_layer_outmap, vec_layer_outmatrix.data(), layer_outmap_rows, layer_outmap_cols);

		}
	}
}

void CUDA_Algo_Lib::CUDACNN::UpdateParas()
{
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.begin();
	iter++;

	CUDA_Algo_Lib::g_idx_itor = 0;//begining at index 0 r_layer
	char str_file_kernel[1000];// initialized properly
	char str_file_bias[1000];// initialized properly

	for (iter; iter < vec_layers_.end(); iter++)
	{
		CUDA_Algo_Lib::g_idx_itor = CUDA_Algo_Lib::g_idx_itor + 1;
		sprintf(str_file_kernel, "./data/kernel_weight/kernel_weight_%d_%d", CUDA_Algo_Lib::g_idx_itor, (*iter).GetType());
		sprintf(str_file_bias, "./data/bias/bias_%d_%d", CUDA_Algo_Lib::g_idx_itor, (*iter).GetType());
		//printf("%s", str_file_kernel);

		switch ((*iter).GetType())
		{
		case 'C':
			UpdateKernels(*iter, *(iter - 1), str_file_kernel, eta_conv_, alpha_conv_);
			UpdateBias(*iter, str_file_bias, eta_conv_);
			break;
		case 'H':
			UpdateKernels(*iter, *(iter - 1), str_file_kernel, eta_fc_, alpha_fc_);
			UpdateBias(*iter, str_file_bias, eta_fc_);
			break;
		case 'O':
			UpdateKernels(*iter, *(iter - 1), str_file_kernel, eta_fc_, alpha_fc_);
			UpdateBias(*iter, str_file_bias, eta_fc_);
			break;
		default:
			break;
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::UpdateBias(CUDA_Algo_Lib::CUDACNNLayer& r_layer, char* str_File_Bias, float eta)
{
	size_t layer_outmap_num = r_layer.GetOutMapNum();
	size_t layer_outmap_rows = r_layer.GetMapSize().rows_;
	size_t layer_outmap_cols = r_layer.GetMapSize().cols_;
	float* p_layer_error = r_layer.vec_errors_.data();
	std::vector<float> vec_error((layer_outmap_rows* layer_outmap_cols), 0.0);
	float deltaBias = 0.0;

	eta = (-1.0) * eta;

	for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
	{

		CalErrorsSum(p_layer_error, idx_layer_outmap, layer_outmap_num, layer_outmap_rows, layer_outmap_cols, batch_size_, vec_error.data());
		deltaBias = (CalErrorSum(vec_error.data(), layer_outmap_rows, layer_outmap_cols) / ((float)batch_size_));
		r_layer.vec_bias_.at(idx_layer_outmap) += (eta * deltaBias);

		/***save bias_***/
		if ((CUDA_Algo_Lib::g_iteration_num - 1) == CUDA_Algo_Lib::g_idx_iteration_num) {
			char str_file_bias_1[1000];
			sprintf(str_file_bias_1, "%s_%d.txt", str_File_Bias, idx_layer_outmap);
			FILE* fp_bias = fopen(str_file_bias_1, "w");

			fprintf(fp_bias, "%f ", r_layer.vec_bias_.at(idx_layer_outmap));
			fprintf(fp_bias, "\n");

			fclose(fp_bias);
		}
	}

}

void CUDA_Algo_Lib::CUDACNN::UpdateKernels(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer, char* str_File_Kernel, float eta, float alpha)
{
	size_t lastlayer_outmap_num = r_lastlayer.GetOutMapNum();
	size_t lastlayer_outmap_rows = r_lastlayer.GetMapSize().rows_;
	size_t lastlayer_outmap_cols = r_lastlayer.GetMapSize().cols_;
	size_t layer_outmap_num = r_layer.GetOutMapNum();
	size_t layer_outmap_rows = r_layer.GetMapSize().rows_;
	size_t layer_outmap_cols = r_layer.GetMapSize().cols_;
	size_t layer_kernel_rows = r_layer.GetKernelSize().rows_;
	size_t layer_kernel_cols = r_layer.GetKernelSize().cols_;


	std::vector<float> vec_delta_kernel_1((layer_kernel_rows * layer_kernel_cols), 0.0);
	std::vector<float> vec_delta_kernel_2((layer_kernel_rows * layer_kernel_cols), 0.0);
	std::vector<float> vec_delta_now((layer_kernel_rows * layer_kernel_cols), 0.0);
	float* p_layer_error = nullptr;
	float* p_lastlayer_outmap = nullptr;
	float* p_layer_laststep_delta_kernel = nullptr;
	float* p_layer_kernel = nullptr;

	eta = (-1.0) * eta;
	alpha = (-1.0) * alpha;

	for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
	{
		for (size_t idx_lastlayer_outmap = 0; idx_lastlayer_outmap < lastlayer_outmap_num; idx_lastlayer_outmap++)
		{
			for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
			{
				p_layer_error = r_layer.GetError(idx_batch, idx_layer_outmap);
				if (idx_batch == 0) {
					size_t shift_idx_lastlayer_batch_map = idx_batch * lastlayer_outmap_num * lastlayer_outmap_rows * lastlayer_outmap_cols;
					size_t shift_idx_lastlayer_out_map = idx_lastlayer_outmap * lastlayer_outmap_rows * lastlayer_outmap_cols;
					p_lastlayer_outmap = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;
					ConvNValid(p_lastlayer_outmap, p_layer_error, lastlayer_outmap_rows, lastlayer_outmap_cols, layer_outmap_rows, layer_outmap_cols, vec_delta_kernel_1.data());
				}
				else {
					size_t shift_idx_lastlayer_batch_map = idx_batch * lastlayer_outmap_num * lastlayer_outmap_rows * lastlayer_outmap_cols;
					size_t shift_idx_lastlayer_out_map = idx_lastlayer_outmap * lastlayer_outmap_rows * lastlayer_outmap_cols;
					p_lastlayer_outmap = r_lastlayer.vec_output_maps_.data() + shift_idx_lastlayer_batch_map + shift_idx_lastlayer_out_map;
					ConvNValid(p_lastlayer_outmap, p_layer_error, lastlayer_outmap_rows, lastlayer_outmap_cols, layer_outmap_rows, layer_outmap_cols, vec_delta_now.data());
					CalConvArrayPlus(vec_delta_now.data(), vec_delta_kernel_1.data(), layer_kernel_rows, layer_kernel_cols);
				}
			}
			size_t shift_idx_layer_kernel_lastlayer = idx_lastlayer_outmap * layer_outmap_num * layer_kernel_rows * layer_kernel_cols;
			size_t shift_idx_layer_kernel_layer = idx_layer_outmap * layer_kernel_rows * layer_kernel_cols;
			p_layer_laststep_delta_kernel = r_layer.vec_laststep_delta_kernel_.data() + shift_idx_layer_kernel_lastlayer + shift_idx_layer_kernel_layer;
			p_layer_kernel = r_layer.vec_kernel_.data() + shift_idx_layer_kernel_lastlayer + shift_idx_layer_kernel_layer;
			SetKernelValue(vec_delta_kernel_2.data(), p_layer_laststep_delta_kernel, layer_kernel_rows, layer_kernel_cols);
			CalArrayMultiply(vec_delta_kernel_2.data(), alpha, layer_kernel_rows, layer_kernel_cols);//for adding momentum
			CalArrayPlus(vec_delta_kernel_2.data(), p_layer_kernel, layer_kernel_rows, layer_kernel_cols);//for adding momentum
			CalArrayDivide(vec_delta_kernel_1.data(), batch_size_, layer_kernel_rows, layer_kernel_cols);
			CalArrayMultiply(vec_delta_kernel_1.data(), eta, layer_kernel_rows, layer_kernel_cols);
			SetKernelValue(p_layer_laststep_delta_kernel, vec_delta_kernel_1.data(), layer_kernel_rows, layer_kernel_cols);//for adding momentum
			CalArrayPlus(vec_delta_kernel_1.data(), p_layer_kernel, layer_kernel_rows, layer_kernel_cols);

			/***save kernel_ weight***/
			if ((CUDA_Algo_Lib::g_iteration_num - 1) == CUDA_Algo_Lib::g_idx_iteration_num) {
				char str_file_kernel_1[1000];
				sprintf(str_file_kernel_1, "%s_%d_%d.txt", str_File_Kernel, idx_lastlayer_outmap, idx_layer_outmap);

				FILE* fp = fopen(str_file_kernel_1, "w");
				size_t shift_idx_layer_kernel_lastlayer = 0;
				size_t shift_idx_layer_kernel_layer = 0;
				size_t idx_layer_kernel = 0;

				for (size_t mm = 0; mm < layer_kernel_rows; mm++)
				{
					for (size_t nn = 0; nn < layer_kernel_cols; nn++)
					{
						shift_idx_layer_kernel_lastlayer = idx_lastlayer_outmap * layer_outmap_num * layer_kernel_rows * layer_kernel_cols;
						shift_idx_layer_kernel_layer = idx_layer_outmap * layer_kernel_rows * layer_kernel_cols;
						idx_layer_kernel = shift_idx_layer_kernel_lastlayer + shift_idx_layer_kernel_layer + (mm * layer_kernel_cols + nn);

						fprintf(fp, "%f ", r_layer.vec_kernel_.at(idx_layer_kernel));
					}

				}
				fprintf(fp, "\n");
				fclose(fp);
			}

		}
	}

}

void CUDA_Algo_Lib::CUDACNN::LoadParas()
{
	CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.begin();
	iter++;

	CUDA_Algo_Lib::g_idx_itor = 0;//begining at index 0 r_layer
	char str_file_kernel[1000];// initialized properly
	char str_file_bias[1000];// initialized properly

	for (iter; iter < vec_layers_.end(); iter++)
	{
		CUDA_Algo_Lib::g_idx_itor = CUDA_Algo_Lib::g_idx_itor + 1;
		sprintf(str_file_kernel, "./data/kernel_weight/kernel_weight_%d_%d", CUDA_Algo_Lib::g_idx_itor, (*iter).GetType());
		sprintf(str_file_bias, "./data/bias/bias_%d_%d", CUDA_Algo_Lib::g_idx_itor, (*iter).GetType());
		//printf("%s", str_file_kernel);

		switch ((*iter).GetType())
		{
		case 'C':
			LoadKernels(*iter, *(iter - 1), str_file_kernel);
			LoadBias(*iter, str_file_bias);
			break;
		case 'H':
			LoadKernels(*iter, *(iter - 1), str_file_kernel);
			LoadBias(*iter, str_file_bias);
			break;
		case 'O':
			LoadKernels(*iter, *(iter - 1), str_file_kernel);
			LoadBias(*iter, str_file_bias);
			break;
		default:
			break;
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::LoadBias(CUDA_Algo_Lib::CUDACNNLayer& r_layer, char* str_File_Bias)
{
	size_t layer_outmap_num = r_layer.GetOutMapNum();
	float bias = 0.0;

	for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
	{
		bias = 0.0;
		/***load bias***/
		char str_file_bias_1[1000];
		sprintf(str_file_bias_1, "%s_%d.txt", str_File_Bias, idx_layer_outmap);
		printf("%s\n", str_file_bias_1);
		FILE* fp_bias = fopen(str_file_bias_1, "r");
		fscanf(fp_bias, "%f ", &bias);
		fclose(fp_bias);

		r_layer.vec_bias_.at(idx_layer_outmap) = bias;
		printf("bias: %f\n", r_layer.vec_bias_.at(idx_layer_outmap));
	}
	
}

void CUDA_Algo_Lib::CUDACNN::LoadKernels(CUDA_Algo_Lib::CUDACNNLayer& r_layer, CUDA_Algo_Lib::CUDACNNLayer& r_lastlayer, char* str_File_Kernel)
{
	
	const size_t lastlayer_outmap_num = r_lastlayer.GetOutMapNum();
	const size_t lastlayer_outmap_rows = r_lastlayer.GetMapSize().rows_;
	const size_t lastlayer_outmap_cols = r_lastlayer.GetMapSize().cols_;
	const size_t layer_outmap_num = r_layer.GetOutMapNum();
	const size_t layer_outmap_rows = r_layer.GetMapSize().rows_;
	const size_t layer_outmap_cols = r_layer.GetMapSize().cols_;
	const size_t layer_kernel_rows = r_layer.GetKernelSize().rows_;
	const size_t layer_kernel_cols = r_layer.GetKernelSize().cols_;

	size_t shift_idx_layer_kernel_lastlayer = 0;
	size_t shift_idx_layer_kernel_layer = 0;
	size_t idx_layer_kernel = 0;

	std::vector<float> vec_kernel((layer_kernel_rows * layer_kernel_cols), 0.0);

	for (size_t idx_layer_outmap = 0; idx_layer_outmap < layer_outmap_num; idx_layer_outmap++)
	{
		for (size_t idx_lastlayer_outmap = 0; idx_lastlayer_outmap < lastlayer_outmap_num; idx_lastlayer_outmap++)
		{
			/***load kernel_ weight***/
			char str_file_kernel_1[1000];
			sprintf(str_file_kernel_1, "%s_%d_%d.txt", str_File_Kernel, idx_lastlayer_outmap, idx_layer_outmap);
			printf("%s\n", str_file_kernel_1);
			FILE* fp_kernel = fopen(str_file_kernel_1, "r");

			for (size_t mm = 0; mm < layer_kernel_rows; mm++)
			{
				for (size_t nn = 0; nn < layer_kernel_cols; nn++)
				{
					shift_idx_layer_kernel_lastlayer = idx_lastlayer_outmap * layer_outmap_num * layer_kernel_rows * layer_kernel_cols;
					shift_idx_layer_kernel_layer = idx_layer_outmap * layer_kernel_rows * layer_kernel_cols;
					idx_layer_kernel = shift_idx_layer_kernel_lastlayer + shift_idx_layer_kernel_layer + (mm * layer_kernel_cols + nn);

					fscanf(fp_kernel, "%f ", (vec_kernel.data()+(mm * layer_kernel_cols + nn)));
					r_layer.vec_kernel_.at(idx_layer_kernel) = vec_kernel.at(mm * layer_kernel_cols + nn);
					printf("kernel_: %f\n", r_layer.vec_kernel_.at(idx_layer_kernel));
				}

			}
			fclose(fp_kernel);
		}
	}
}

void CUDA_Algo_Lib::CUDACNN::Inference(CUDA_Algo_Lib::DatasetLoadingParamPKG& r_dataset_param)
{
	std::cout << "Start Inference" << std::endl;

	size_t total_false = 0, false_1 = 0, false_2 = 0, predict, real;
	size_t total_num_iter = r_dataset_param.total_num_images_ / batch_size_;

	float* p_inference_batch_data = nullptr;
	float* p_inference_batch_label = nullptr;
	std::vector<float> vec_inference_batch_data;
	std::vector<float> vec_inference_batch_label;
	vec_inference_batch_data.reserve(batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_);
	vec_inference_batch_data.resize(batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_);
	vec_inference_batch_label.reserve(batch_size_ * r_dataset_param.num_output_cls_);
	vec_inference_batch_label.resize(batch_size_ * r_dataset_param.num_output_cls_);

	FILE* p_file_error_predict_neg = fopen("./outputdata/error_predict_neg_filename.txt", "w");
	FILE* p_file_error_predict_pos = fopen("./outputdata/error_predict_pos_filename.txt", "w");
	for (size_t idx_iteration = 0; idx_iteration < total_num_iter; idx_iteration++)
	{
		std::cout << "NO.of iteration(testing): " << idx_iteration << std::endl;
		size_t idx_inference_dataset_batch = idx_iteration % (r_dataset_param.total_num_images_ / batch_size_);
		for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
		{
			std::cout << "NO.of batch(testing): " << idx_batch << std::endl;
		
			std::vector<float>::iterator shift_begin_iter_loaded_dataset_batch_data;
			std::vector<float>::iterator shift_begin_iter_loaded_dataset_batch_label;
			shift_begin_iter_loaded_dataset_batch_data = r_dataset_param.vec_images_.begin() + (idx_inference_dataset_batch * batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_);
			shift_begin_iter_loaded_dataset_batch_label = r_dataset_param.vec_labels_.begin() + (idx_inference_dataset_batch * batch_size_ * r_dataset_param.num_output_cls_);
			vec_inference_batch_data.assign(shift_begin_iter_loaded_dataset_batch_data, (shift_begin_iter_loaded_dataset_batch_data + (batch_size_ * r_dataset_param.channels_image_ * r_dataset_param.rows_image_ * r_dataset_param.cols_image_)));
			vec_inference_batch_label.assign(shift_begin_iter_loaded_dataset_batch_label, (shift_begin_iter_loaded_dataset_batch_label + (batch_size_ * r_dataset_param.num_output_cls_)));

		}

		Forward(vec_inference_batch_data.data());
		CUDA_Algo_Lib::VECCUDACNNLayers::iterator iter = vec_layers_.end();
		iter--;
		for (size_t idx_batch = 0; idx_batch < batch_size_; idx_batch++)
		{
			std::cout << idx_batch << std::endl;

			size_t layer_outmap_num = (*iter).GetOutMapNum();
			size_t layer_outmap_rows = (*iter).GetMapSize().rows_;
			size_t layer_outmap_cols = (*iter).GetMapSize().cols_;
			size_t shift_idx_layer_batch_map = idx_batch * layer_outmap_num * layer_outmap_rows * layer_outmap_cols;
			float* p_layer_batchmap = (*iter).vec_output_maps_.data() + shift_idx_layer_batch_map;
			predict = FindIndex(p_layer_batchmap, layer_outmap_num, layer_outmap_rows, layer_outmap_cols);

			float* p_batch_gt_label = vec_inference_batch_label.data() + (idx_batch * r_dataset_param.num_output_cls_);
			real = FindIndex(p_batch_gt_label, r_dataset_param.num_output_cls_);


			//predict For batch size=2
			if (0 == idx_batch) {
				if (predict != real)
				{
					false_1++;
					//num_charaters_neg1 = sprintf(_input_negfilename, "%s%d%s", _negfilepath, idx_iteration, _imgfileextension);
					//printf("error predict-number of charaters: %d, string: \"%s\"\n", num_charaters_neg1, _input_negfilename);
					//fprintf(p_file_error_predict_neg, "%s\n", _input_negfilename);

				}
			}
			else if (1 == idx_batch) {
				if (predict != real)
				{
					false_2++;
					//num_charaters_pos1 = sprintf(_input_posfilename, "%s%d%s", _posfilepath, idx_iteration, _imgfileextension);
					//num_charaters_pos1 = sprintf(_input_posfilename, "%s%d", _posfilepath, idx_iteration);
					//printf("error predict-number of charaters: %d, string: \"%s\"\n", num_charaters_pos1, _input_posfilename);
					//fprintf(p_file_error_predict_pos, "%s\n", _input_posfilename);
				}
			}
		}
	}

	total_false = false_1 + false_2;

	std::cout << "+++++++Finish Inference+++++++" << std::endl;
	std::cout << "Error predict number of neg: " << false_1 << std::endl;
	std::cout << "Error rate of neg: " << (float)false_1 / (float)r_dataset_param.num_neg_images_ << std::endl;
	std::cout << "Error predict number of pos: " << false_2 << std::endl;
	std::cout << "Error rate of pos: " << (float)false_2 / (float)r_dataset_param.num_pos_images_ << std::endl;
	std::cout << "Error predict total number: " << total_false << std::endl;
	std::cout << "Total error rate: " << (float)total_false / (float)r_dataset_param.total_num_images_ << std::endl << std::endl;

	FILE* p_file_false_metrics;
	p_file_false_metrics = fopen("./outputdata/false_metrics.txt", "a");
	/*
	if( (err=fopen_s(&p_file_false_metrics, "fausePrun.txt", "a")) != 0 )
		exit(1) ;
	*/
	CUDA_Algo_Lib::g_idx_epoch++;
	fprintf(p_file_false_metrics, "epoch: %4d\n", CUDA_Algo_Lib::g_idx_epoch);
	fprintf(p_file_false_metrics, "neg: %4d %8f\n", false_1, (float)false_1 / (float)r_dataset_param.num_neg_images_);
	fprintf(p_file_false_metrics, "pos: %4d %8f\n", false_2, (float)false_2 / (float)r_dataset_param.num_pos_images_);
	fprintf(p_file_false_metrics, "total: %4d %8f\n\n", total_false, (float)total_false / (float)r_dataset_param.total_num_images_);
	fclose(p_file_false_metrics);
	fclose(p_file_error_predict_pos);
	fclose(p_file_error_predict_neg);

}
